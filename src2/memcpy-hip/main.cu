#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define NUM_SIZE 16

void setup(size_t *size) {
  for (int i = 0; i < NUM_SIZE; i++) {
    size[i] = 1 << (i + 6);
  }
}

void valSet(int* A, int val, size_t size) {
  size_t len = size / sizeof(int);
  for (size_t i = 0; i < len; i++) {
    A[i] = val;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    std::cerr << "Usage: " << argv[0] << " <repeat>\n";
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int *d_A;
  size_t size[NUM_SIZE];
  hipError_t err;

  setup(size);

  for (int i = 0; i < NUM_SIZE; i++) {
    int* A = (int*)malloc(size[i]);
    if (A == nullptr) {
      std::cerr << "Host memory allocation failed\n";
      return -1;
    }	
    valSet(A, 1, size[i]);


    err = hipMalloc((void**)&d_A, size[i]);
    if (err != hipSuccess) {
      std::cerr << "Device memory allocation failed\n";
      free(A);
      return -1;
    }

    // warmup
    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(d_A, A, size[i], hipMemcpyHostToDevice, 0);
    }
    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(d_A, A, size[i], hipMemcpyHostToDevice, 0);
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();
    auto timeH2D = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    std::cout << "Copy " << size[i] << " bytes from host to device takes "
              << (timeH2D * 1e-3f) / repeat <<  " us" << std::endl;

    // warmup
    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(A, d_A, size[i], hipMemcpyDeviceToHost, 0);
    }
    hipDeviceSynchronize();

    start = std::chrono::steady_clock::now();

    for (int j = 0; j < repeat; j++) {
      hipMemcpyAsync(A, d_A, size[i], hipMemcpyDeviceToHost, 0);
    }
    hipDeviceSynchronize();

    end = std::chrono::steady_clock::now();
    auto timeD2H = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    std::cout << "Copy " << size[i] << " bytes from device to host takes "
              << (timeD2H * 1e-3f) / repeat <<  " us" << std::endl;

    hipFree(d_A);
    free(A);
    std::cout << "Timing gap in nanoseconds per byte: "
              << (float)std::abs(timeH2D - timeD2H) / (repeat * size[i]);
    std::cout << std::endl << std::endl;
  }
  return 0;
}
