#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <chrono>
#include <random>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

__global__ 
void filter (int *__restrict__ dst,
             int *__restrict__ nres,
             const int*__restrict__ src,
             int n)
{
  __shared__ int l_n;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // zero the counter
  if (threadIdx.x == 0)
    l_n = 0;
  __syncthreads();

  // get the value, evaluate the predicate, and
  // increment the counter if needed
  int d, pos;

  if(i < n) {
    d = src[i];
    if(d > 0)
      pos = atomicAdd(&l_n, 1);
  }
  __syncthreads();

  // leader increments the global counter
  if(threadIdx.x == 0)
    l_n = atomicAdd(nres, l_n);
  __syncthreads();

  // threads with true predicates write their elements
  if(i < n && d > 0) {
    pos += l_n; // increment local pos by global counter
    dst[pos] = d;
  }
  __syncthreads();
}


__device__ int atomicAggInc(int *ctr) {
  auto g = cooperative_groups::coalesced_threads();
  int warp_res = 0;
  if(g.thread_rank() == 0)
    warp_res = atomicAdd(ctr, g.size());
  return g.shfl(warp_res, 0) + g.thread_rank();
}

__global__
void filter2 (int *__restrict__ dst,
              int *__restrict__ nres,
              const int*__restrict__ src,
              int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < n && src[i] > 0)
    dst[atomicAggInc(nres)] = src[i];
}


// compare device results with host results
bool check(int *d_nres, int *d_output, int h_nres, std::vector<int> &h_output) {
  int nres;
  hipMemcpy(&nres, d_nres, sizeof(int), hipMemcpyDeviceToHost);

  std::vector<int> output (nres);

  hipMemcpy(output.data(), d_output, sizeof(int) * nres, hipMemcpyDeviceToHost);

  // clear device output
  hipMemset(d_output, 0, sizeof(int) * nres);

  std::sort(output.begin(), output.end());

  bool equal = (h_nres == nres) && 
               std::equal(h_output.begin(),
                          h_output.begin() + h_nres, output.begin());
  return equal;
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("Usage: %s <number of elements> <block size> <repeat>\n", argv[0]);
    return 1;
  }
  const int num_elems = atoi(argv[1]);
  const int block_size = atoi(argv[2]);
  const int repeat = atoi(argv[3]);
    
  std::vector<int> input (num_elems);

  // Generate input data.
  for (int i = 0; i < num_elems; i++) {
    input[i] = i - num_elems / 2;
  }

  std::mt19937 g;
  g.seed(19937);
  std::shuffle(input.begin(), input.end(), g);

  // Generate host output with host filtering code.
  std::vector<int> h_output (num_elems);

  int h_flt_count = 0;
  for (int i = 0; i < num_elems; i++) {
    if (input[i] > 0) {
      h_output[h_flt_count++] = input[i];
    }
  }
  // Sort the result for comparison
  std::sort(h_output.begin(), h_output.begin() + h_flt_count);

  // Filtering on a device
  int *d_input, *d_output, *d_nres;

  hipMalloc(&d_input, sizeof(int) * num_elems);
  hipMalloc(&d_output, sizeof(int) * num_elems);
  hipMalloc(&d_nres, sizeof(int));

  hipMemcpy(d_input, input.data(),
             sizeof(int) * num_elems, hipMemcpyHostToDevice);

  dim3 dimBlock (block_size);
  dim3 dimGrid ((num_elems + block_size - 1) / block_size);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    hipMemset(d_nres, 0, sizeof(int));
    filter<<<dimGrid, dimBlock>>>(d_output, d_nres, d_input, num_elems);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of filter (shared memory) %lf (ms)\n",
         (time * 1e-6) / repeat);

  bool match = check(d_nres, d_output, h_flt_count, h_output);
  printf("%s\n", match ? "PASS" : "FAIL");

  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    hipMemset(d_nres, 0, sizeof(int));
    filter2<<<dimGrid, dimBlock>>>(d_output, d_nres, d_input, num_elems);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of filter (global aggregate) %lf (ms)\n",
         (time * 1e-6) / repeat);

  match = check(d_nres, d_output, h_flt_count, h_output);
  printf("%s\n", match ? "PASS" : "FAIL");

  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_nres);

  return 0;
}
