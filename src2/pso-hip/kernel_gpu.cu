#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "kernel.h"

__device__ float fitness_function(float x[])
{
  float y1 = F(x[0]);
  float yn = F(x[DIM-1]);
  float res = powf(sinf(phi*y1), 2.f) + powf(yn-1, 2.f);

  for(int i = 0; i < DIM-1; i++)
  {
    float y = F(x[i]);
    float yp = F(x[i+1]);
    res += powf(y-1.f, 2.f) * (1.f + 10.f * powf(sinf(phi*yp), 2.f));
  }

  return res;
}

__global__
void kernelUpdateParticle(float *__restrict__ positions,
                          float *__restrict__ velocities,
                          const float *__restrict__ pBests,
                          const float *__restrict__ gBest,
                          const int p,
                          const float rp,
                          const float rg)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= p*DIM) return;

  velocities[i]=OMEGA*velocities[i]+
                c1*rp*(pBests[i]-positions[i])+
                c2*rg*(gBest[i%DIM]-positions[i]);
  positions[i]+=velocities[i];
}

__global__
void kernelUpdatePBest(const float *__restrict__ positions,
                             float *__restrict__ pBests,
                             float *__restrict__ gBest,
                       const int p)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= p) return;
  i = i*DIM;

  float tempParticle1[DIM];
  float tempParticle2[DIM];

  for(int j=0;j<DIM;j++)
  {
    tempParticle1[j]=positions[i+j];
    tempParticle2[j]=pBests[i+j];
  }

  if(fitness_function(tempParticle1)<fitness_function(tempParticle2))
  {
    for(int j=0;j<DIM;j++)
      pBests[i+j]=tempParticle1[j];

    if(fitness_function(tempParticle1)<130.f) //fitness_function(gBest))
    {
      for(int j=0;j<DIM;j++) {
        atomicAdd(gBest+j,tempParticle1[j]);
      }
    }
  }
}

extern "C" void gpu_pso(int p, int r,
                        float *positions,float *velocities,float *pBests,float *gBest)
{
  int size = p*DIM;
  size_t size_byte = sizeof(float) * size;
  size_t res_size_byte = sizeof(float) * DIM;

  float *devPos;
  float *devVel;
  float *devPBest;
  float *devGBest;

  hipMalloc((void**)&devPos,size_byte);
  hipMalloc((void**)&devVel,size_byte);
  hipMalloc((void**)&devPBest,size_byte);
  hipMalloc((void**)&devGBest,res_size_byte);

  int threadNum=256;
  int blocksNum1=(size+threadNum-1)/threadNum;
  int blocksNum2=(p+threadNum-1)/threadNum;

  hipMemcpy(devPos,positions,size_byte,hipMemcpyHostToDevice);
  hipMemcpy(devVel,velocities,size_byte,hipMemcpyHostToDevice);
  hipMemcpy(devPBest,pBests,size_byte,hipMemcpyHostToDevice);
  hipMemcpy(devGBest,gBest,res_size_byte,hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for(int iter=0;iter<r;iter++)
  {
    float rp=getRandomClamped(iter);
    float rg=getRandomClamped(r-iter);
    hipLaunchKernelGGL(kernelUpdateParticle, blocksNum1, threadNum, 0, 0, 
      devPos,devVel,devPBest,devGBest,
      p,rp,rg);

    hipLaunchKernelGGL(kernelUpdatePBest, blocksNum2, threadNum, 0, 0, devPos,devPBest,devGBest,p);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time %f (us)\n", time * 1e-3f / r);
  
  hipMemcpy(gBest,devGBest,res_size_byte,hipMemcpyDeviceToHost);
  hipMemcpy(pBests,devPBest,size_byte,hipMemcpyDeviceToHost);

  hipFree(devPos);
  hipFree(devVel);
  hipFree(devPBest);
  hipFree(devGBest);
}
