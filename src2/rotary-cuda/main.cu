#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <thrust/tuple.h>

#define C10_WARP_SIZE 32
#define C10_HOST_DEVICE __host__ __device__

#include "Array.h"
#include "FunctionTraits.h"

#ifndef GPU_LAMBDA
#define GPU_LAMBDA __host__ __device__
#endif

constexpr int num_threads() {
  return C10_WARP_SIZE * 4;
}

constexpr int thread_work_size() { return 4; }
constexpr int block_work_size() { return thread_work_size() * num_threads(); }

template<int arg_index>
struct unroll_load_helper {
  template <typename args_t, typename policy_t, typename offset_t, typename loader_t>
  static __device__ void apply(policy_t &self, args_t *args, offset_t offset, loader_t loader, int j, int num_outputs) {
    using arg_t = std::tuple_element_t<arg_index, args_t>;
    // `data` hold the data_ptr for tensors [output, input0, input1, ...], so we
    // need a+1 offset to get the input
    std::get<arg_index>(args[j]) = loader.template load<arg_t>(self.data[arg_index + num_outputs], offset[arg_index], arg_index);
  }
};

template <int current>
struct multi_outputs_store_helper {
  template<int ntensors, int num_outputs, typename ...Args>
  C10_HOST_DEVICE static void apply(
      Array<char*, ntensors> data,
      Array<uint32_t, num_outputs> offsets,
      thrust::tuple<Args...> ret) {
    using T = typename thrust::tuple_element<current, thrust::tuple<Args...>>::type;
    T *to = reinterpret_cast<T *>(data[current]) + offsets[current];
    *to = thrust::get<current>(ret);
  }
};

// What does the `static_unroll` do?
//
// We want to do something like:
//
//    using args_t = typename traits::ArgsTuple;
//    args_t args;
//    #pragma unroll
//    for (int i = 0; i < traits::arity; i++) {
//      std::get<i>(args) = ....
//    }
//
// but unfortunately the above code does not work because
// the template argument has to be a compile time constant
// so `static_unroll` is created to simulate `#pragma unroll`
// using template metaprogramming.

template<template<int i> typename func, int end, int current=0>
struct static_unroll {
  template<typename... Args>
  static inline C10_HOST_DEVICE void with_args(Args&&... args) {
    func<current>::apply(std::forward<Args>(args)...);
    static_unroll<func, end, current+1>::with_args(args...);
  }
};

template<template<int i> typename func, int end>
struct static_unroll<func, end, end> {
  template<typename... Args>
  static inline C10_HOST_DEVICE void with_args(Args... args) {}
};

template <typename T>
struct LoadImpl {
  C10_HOST_DEVICE static T apply(const void* src) {
    return *reinterpret_cast<const T*>(src);
  }
};

template <>
struct LoadImpl<bool> {
  C10_HOST_DEVICE static bool apply(const void* src) {
    static_assert(sizeof(bool) == sizeof(char));
    // NOTE: [Loading boolean values]
    // Protect against invalid boolean values by loading as a byte
    // first, then converting to bool (see gh-54789).
    return *reinterpret_cast<const unsigned char*>(src);
  }
};

template <typename T>
C10_HOST_DEVICE T load_impl(const void* src) {
  return LoadImpl<T>::apply(src);
}

template <typename scalar_t>
C10_HOST_DEVICE scalar_t load_impl(const scalar_t* src) {
  return LoadImpl<scalar_t>::apply(src);
}

struct LoadWithoutCast {
  template<typename scalar_t>
  __device__ scalar_t load(char *base_ptr, uint32_t offset, int arg) {
    return load_impl(reinterpret_cast<scalar_t *>(base_ptr) + offset);
  }
};

template <typename data_t, typename inp_calc_t, typename out_calc_t, int num_outputs>
struct multi_outputs_unroll {
  //multi_outputs_unroll struct members and check_inbounds and load methods are copypasted from unroll struct
  //we don't use inheritance because of compiler bug in cuda 10.2+
  data_t data;
  int remaining;
  inp_calc_t input_offset_calculator;
  out_calc_t output_offset_calculator;
  LoadWithoutCast loader;

  __device__ multi_outputs_unroll(data_t data, int remaining, inp_calc_t ic, out_calc_t oc):
    data(data), remaining(remaining), input_offset_calculator(ic), output_offset_calculator(oc) {}

  __device__ inline bool check_inbounds(int thread_work_elem) {
    return ((threadIdx.x  + thread_work_elem*num_threads()) < remaining);
  }

  template<typename args_t>
  __device__ inline void load(args_t *args, int idx) {
    constexpr int arity = std::tuple_size<args_t>::value;
    int thread_idx = threadIdx.x;
    #pragma unroll
    for (int i = 0; i < thread_work_size(); i++) {
      if (thread_idx >= remaining) {
        return;
      }
      int linear_idx = thread_idx + block_work_size() * idx;
      auto offset = input_offset_calculator.get(linear_idx);
      static_unroll<unroll_load_helper, arity>::with_args(*this, args, offset, loader, i, num_outputs);
      thread_idx += num_threads();
    }
  }

  template <typename return_t>
  __device__ inline void store(return_t *from, int idx) {
    int thread_idx = threadIdx.x;
    #pragma unroll
    for (int i = 0; i < thread_work_size(); i++) {
      if (thread_idx >= this->remaining) {
        return;
      }
      int linear_idx = thread_idx + block_work_size() * idx;
      auto offsets = this->output_offset_calculator.get(linear_idx);
      static_unroll<multi_outputs_store_helper, num_outputs>::with_args(this->data, offsets, from[i]);
      thread_idx += num_threads();
    }
  }
};

template <class F, class Tuple, std::size_t... INDEX>
// GCC/Clang need the decltype() return type
C10_HOST_DEVICE constexpr decltype(auto) apply_impl(
    F&& f,
    Tuple&& t,
    std::index_sequence<INDEX...>)
{
  return std::forward<F>(f)(std::get<INDEX>(std::forward<Tuple>(t))...);
}

template <class F, class Tuple>
C10_HOST_DEVICE constexpr decltype(auto) guts_apply(F&& f, Tuple&& t) {
  return apply_impl(
      std::forward<F>(f),
      std::forward<Tuple>(t),
      std::make_index_sequence<
          std::tuple_size<std::remove_reference_t<Tuple>>::value>{});
}


template<typename func_t, typename policy_t>
__device__ inline void elementwise_kernel_helper(func_t f, policy_t policy) {
  using traits = function_traits<func_t>;
  using return_t = typename traits::result_type;
  using args_t = typename traits::ArgsTuple;

  int idx = blockIdx.x;

  return_t results[thread_work_size()];
  args_t args[thread_work_size()];

  // load
  policy.load(args, idx);

  // compute
  #pragma unroll
  for (int i = 0; i < thread_work_size(); i++) {
    if (policy.check_inbounds(i)) {
      results[i] = guts_apply(f, args[i]);
    }
  }

  // store
  policy.store(results, idx);
}

template <int num_outputs, typename func_t, typename array_t, typename inp_calc_t, typename out_calc_t>
__global__ void unrolled_elementwise_kernel_for_multi_outputs(int N, func_t f, array_t data, inp_calc_t ic, out_calc_t oc) {
  int remaining = N - block_work_size() * blockIdx.x;
  elementwise_kernel_helper(f, multi_outputs_unroll<array_t, inp_calc_t, out_calc_t, num_outputs>(data, remaining, ic, oc));
}

template <int num_outputs, typename func_t, typename array_t, typename inp_calc_t, typename out_calc_t>
static inline void launch_unrolled_kernel_for_multi_outputs(int64_t N, const func_t& f, array_t data, inp_calc_t ic, out_calc_t oc) {
  int64_t grid = (N + block_work_size() - 1) / block_work_size();
  unrolled_elementwise_kernel_for_multi_outputs<num_outputs, func_t, array_t><<<grid, num_threads()>>>(N, f, data, ic, oc);
}

template <int NARGS, typename index_t = uint32_t>
struct TrivialOffsetCalculator {
  // The offset for each argument. Wrapper around fixed-size array.
  // The offsets are in # of elements, not in bytes.
  // On CUDA, zero sized array is not allowed, so when we are handling nullary
  // operators, we need to create a size 1 offset to avoid compiler failure.
  // This size 1 offset is just a placeholder, and we will not use it.
  using offset_type = Array<index_t, std::max<int>(NARGS, 1)>;

  C10_HOST_DEVICE offset_type get(index_t linear_idx) const {
    offset_type offsets;
    #pragma unroll
    for (int arg = 0; arg < NARGS; arg++) {
      offsets[arg] = linear_idx;
    }
    return offsets;
  }
};

template <typename func_t>
void gpu_kernel_multiple_outputs_impl(const int repeat, const func_t& f) {
  constexpr int num_outputs = 2;
  constexpr int num_inputs = 4;
  constexpr int ntensors = num_outputs + num_inputs;

  int64_t numel = block_work_size() * 10000;
  printf("Number of elements: %zu\n", numel);

  uint64_t size = numel * sizeof(float);
  
  float *h_x1 = (float*) malloc (size);
  float *h_x2 = (float*) malloc (size);
  float *h_cos = (float*) malloc (size);
  float *h_sin = (float*) malloc (size);
  float *h_o1 = (float*) malloc (size);
  float *h_o2 = (float*) malloc (size);
  for (int64_t i = 0; i < numel; i++) {
    h_x1[i] = 1.f * (i+1) / numel;
    h_x2[i] = 1.f * (i+1) / numel;
    h_cos[i] = cosf(i / powf(10000, i / numel)); 
    h_sin[i] = sinf(i / powf(10000, i / numel));
  }
  
  float *d_x1, *d_x2, *d_cos, *d_sin, *d_o1, *d_o2;
  hipMalloc((void**)&d_x1, size); 
  hipMalloc((void**)&d_x2, size);
  hipMalloc((void**)&d_cos, size);
  hipMalloc((void**)&d_sin, size);
  hipMalloc((void**)&d_o1, size);
  hipMalloc((void**)&d_o2, size);

  hipMemcpy(d_x1, h_x1, size, hipMemcpyHostToDevice);
  hipMemcpy(d_x2, h_x2, size, hipMemcpyHostToDevice);
  hipMemcpy(d_cos, h_cos, size, hipMemcpyHostToDevice);
  hipMemcpy(d_sin, h_sin, size, hipMemcpyHostToDevice);

  Array<char*, ntensors> data;
  data[0] = (char*)d_o1;
  data[1] = (char*)d_o2;
  data[2] = (char*)d_x1;
  data[3] = (char*)d_x2;
  data[4] = (char*)d_cos;
  data[5] = (char*)d_sin;

  auto input_calc = TrivialOffsetCalculator<num_inputs>();
  auto output_calc = TrivialOffsetCalculator<num_outputs>();

  printf("Number of blocks: %zu, block size: %d\n", 
         (numel + block_work_size() - 1) / block_work_size(), num_threads());

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    launch_unrolled_kernel_for_multi_outputs<num_outputs>(numel, f, data, input_calc, output_calc);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(h_o1, d_o1, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_o2, d_o2, size, hipMemcpyDeviceToHost);
  bool ok = true;
  for (int64_t i = 0; i < numel; i++) {
    float r1 = float(h_x1[i]) * float(h_cos[i]) - float(h_x2[i]) * float(h_sin[i]);
    float r2 = float(h_x1[i]) * float(h_sin[i]) + float(h_x2[i]) * float(h_cos[i]);
    if ((r1 - h_o1[i]) > 1e-3f || (r2 - h_o2[i]) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");
  hipFree(d_x1);
  hipFree(d_x2);
  hipFree(d_cos);
  hipFree(d_sin);
  hipFree(d_o1);
  hipFree(d_o2);
  free(h_x1);
  free(h_x2);
  free(h_cos);
  free(h_sin);
  free(h_o1);
  free(h_o2);
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  typedef float scalar_t;

  gpu_kernel_multiple_outputs_impl(repeat,
                [] GPU_LAMBDA (scalar_t x1, scalar_t x2, scalar_t cos,
                               scalar_t sin) -> thrust::tuple<scalar_t, scalar_t> {
                scalar_t out1 = float(x1) * float(cos) - float(x2) * float(sin);
                scalar_t out2 = float(x1) * float(sin) + float(x2) * float(cos);
                return {out1, out2};
   });
}
