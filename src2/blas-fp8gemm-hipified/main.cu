#include "hip/hip_runtime.h"
#include <chrono>
#include "helper.h"

/// Sample wrapper executing fp8 matmul with hipblasLtMatmul, with addition of per-tensor scaling, amax calculations, and
/// the workspace to support split-K algorithms.
///
/// pointer mode is for alpha and beta is always host, to change it configure the appropriate matmul descriptor
/// attribute matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to
/// change this configure appropriate attribute in the preference handle
void LtFp8Matmul(const int repeat,
                 hipblasLtHandle_t ltHandle,
                 int m,
                 int n,
                 int k,
                 const float *alpha, /* host pointer */
                 const float *beta, /* host pointer */
                 const float *a_scale, /* device pointer */
                 const __hip_fp8_e4m3_fnuz *A,
                 int lda,
                 const float *b_scale, /* device pointer */
                 const __hip_fp8_e4m3_fnuz *B,
                 int ldb,
                 const float *c_scale, /* device pointer */
                 const hip_bfloat16 *C,
                 int ldc,
                 const float *d_scale, /* device pointer */
                 __hip_fp8_e4m3_fnuz *D,
                 float *amax_d, /* device pointer */
                 void *workspace,
                 size_t workspaceSize) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // set scaling factors
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER, &c_scale, sizeof(c_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER, &amax_d, sizeof(amax_d)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    // table of supported type combinations can be found in the documentation: https://docs.nvidia.com/cuda/cublas/index.html#cublasltmatmul
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16BF, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_8F_E4M3, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        printf("no heuristic function available for current configuration\n");
        return;
    }

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++) {
      checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha, A, Adesc,
                                     B, Bdesc, beta,
                                     C, Cdesc,
                                     D, Ddesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("Average hipblasLtMatmul execution time %f (us)\n", (time * 1e-3f) / repeat);

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}


int main(int argc, char *argv[])
{
   if (argc != 2) {
     printf("Usage: %s <repeat>\n", argv[0]);
     return 1;
   }
   const int repeat = atoi(argv[1]);


   TestBench<__hip_fp8_e4m3_fnuz, 
             hip_bfloat16, // hipblasLtMatrixLayoutCreate
             __hip_fp8_e4m3_fnuz,
             float> props(64, 128, 256, 2.0f, 1.0f, 32ULL * 1024 * 1024);

   props.run([&props, repeat] {
        LtFp8Matmul(repeat,
                    props.ltHandle,
                    props.m,
                    props.n,
                    props.k,
                    &props.alpha,
                    &props.beta,
                    props.AscaleDev, props.Adev, props.k,
                    props.BscaleDev, props.Bdev, props.k,
                    props.CscaleDev, props.Cdev, props.m,
                    props.DscaleDev, props.Ddev,
                    props.DamaxDev,
                    props.workspace,
                    props.workspaceSize);
    });

    return 0;
}
