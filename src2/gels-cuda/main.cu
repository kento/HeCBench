/*******************************************************************************
 * Copyright 2022 Intel Corporation.
 *
 * This software and the related documents are Intel copyrighted  materials, and
 * your use of  them is  governed by the  express license  under which  they
 *were provided to you (License).  Unless the License provides otherwise, you
 *may not use, modify, copy, publish, distribute,  disclose or transmit this
 *software or the related documents without Intel's prior written permission.
 *
 * This software and the related documents  are provided as  is,  with no
 *express or implied  warranties,  other  than those  that are  expressly stated
 *in the License.
 *******************************************************************************/

/*
 *  Content:
 *       This example demonstrates use of cublasXgelsBatched
 *       to perform batched calculation of least squares.
 *
 *       The supported floating point data types for matrix data are:
 *           float
 *           double
 *           std::complex<float>
 *           std::complex<double>
 *******************************************************************************/

#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <complex>
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

template <typename data_t, typename real_t = decltype(std::real((data_t)0)),
          bool is_real = std::is_same_v<data_t, real_t>>
int run_gels_batch_example(const int repeat) {
  
  const int64_t m = 5, n = 5, nrhs = 1, lda = m, stride_a = n * lda, ldb = m,
                stride_b = nrhs * ldb, batch_size = 2;

  auto v = [](real_t arg) {
    if constexpr (is_real)
      return arg;
    else
      return data_t{0, arg};
  };

  data_t A[] = {
      v( 1.0), v( 0.0), v( 0.0), v( 0.0), v( 0.0),
      v( 1.0), v( 0.2), v(-0.4), v(-0.4), v(-0.8),
      v( 1.0), v( 0.6), v(-0.2), v( 0.4), v(-1.2),
      v( 1.0), v( 1.0), v(-1.0), v( 0.6), v(-0.8),
      v( 1.0), v( 1.8), v(-0.6), v( 0.2), v(-0.6)
                                                 ,
      v( 0.2), v(-0.4), v(-0.4), v(-0.8), v( 0.0),
      v( 0.4), v( 0.2), v( 0.8), v(-0.4), v( 0.0),
      v( 0.4), v(-0.8), v( 0.2), v( 0.4), v( 0.0),
      v( 0.8), v( 0.4), v(-0.4), v( 0.2), v( 0.0),
      v( 0.0), v( 0.0), v( 0.0), v( 0.0), v( 1.0)
  };

  data_t B[] = {
      v(5.0), v(3.6),  v(-2.2), v(0.8),  v(-3.4),
      v(1.8), v(-0.6), v(0.2),  v(-0.6), v(1.0),
  };

  data_t X[] = {
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  };

  hipblasHandle_t h;
  hipblasStatus_t status;
  status = hipblasCreate(&h);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("> ERROR: cuBLAS initialization failed..\n");
    return (EXIT_FAILURE);
  }

  data_t *A_dev;
  hipMalloc((void**)&A_dev, stride_a * batch_size * sizeof(data_t));

  data_t *B_dev;
  hipMalloc((void**)&B_dev, stride_b * batch_size * sizeof(data_t));

  data_t** ptrA_array = (data_t**) malloc (batch_size * sizeof(data_t*));
  for (int i = 0; i < batch_size; i++) ptrA_array[i] = A_dev + (i * stride_a);

  data_t** ptrB_array = (data_t**) malloc (batch_size * sizeof(data_t*));
  for (int i = 0; i < batch_size; i++) ptrB_array[i] = B_dev + (i * stride_b);

  data_t **ptrA_array_dev, **ptrB_array_dev;
  hipMalloc((void**)&ptrA_array_dev, batch_size * sizeof(data_t*));
  hipMalloc((void**)&ptrB_array_dev, batch_size * sizeof(data_t*));
  hipMemcpy(ptrA_array_dev, ptrA_array, batch_size * sizeof(data_t*), hipMemcpyHostToDevice);
  hipMemcpy(ptrB_array_dev, ptrB_array, batch_size * sizeof(data_t*), hipMemcpyHostToDevice);
  int info;

  long time = 0;
  for (int i = 0; i <= repeat; i++) {
    hipMemcpy(A_dev, A, stride_a * batch_size * sizeof(data_t), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B, stride_b * batch_size * sizeof(data_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    if constexpr (std::is_same_v<data_t, float>)
      status = hipblasSgelsBatched(h, HIPBLAS_OP_N, m, n, nrhs, ptrA_array_dev, lda,
                                  ptrB_array_dev, ldb, &info, NULL, batch_size);
    else if constexpr (std::is_same_v<data_t, double>)
      status = hipblasDgelsBatched(h, HIPBLAS_OP_N, m, n, nrhs, ptrA_array_dev, lda,
                                  ptrB_array_dev, ldb, &info, NULL, batch_size);
    else if constexpr (std::is_same_v<data_t, std::complex<float>>)
      status = hipblasCgelsBatched(h, HIPBLAS_OP_N, m, n, nrhs,
                                  reinterpret_cast<hipComplex *const *>(ptrA_array_dev),
                                  lda,
                                  reinterpret_cast<hipComplex *const *>(ptrB_array_dev),
                                  ldb, &info, NULL, batch_size);
    else if constexpr (std::is_same_v<data_t, std::complex<double>>)
      status = hipblasZgelsBatched(h, HIPBLAS_OP_N, m, n, nrhs,
                                  reinterpret_cast<hipDoubleComplex *const *>(ptrA_array_dev),
                                  lda,
                                  reinterpret_cast<hipDoubleComplex *const *>(ptrB_array_dev),
                                  ldb, &info, NULL, batch_size);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    if (i != 0)
      time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("> ERROR: cublasXgelsBatched() failed with error %s..\n",
             cublasGetStatusString(status));
    }
    // If info=0, the parameters passed to the function are valid
    // If info<0, the parameter in position -info is invalid
    if (info < 0)
     printf("The parameter in position %d is invalid\n", -info);
  }
  printf("Average kernel execution time : %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(B, B_dev, stride_b * batch_size * sizeof(data_t), hipMemcpyDeviceToHost);

  const real_t bound = std::is_same_v<real_t, float> ? 1e-6 : 1e-8;
  bool passed = true;

  printf("Results:\n");
  auto print = [](data_t &v) {
    if constexpr (is_real)
      printf("%6.2f", v);
    else
      printf("<%6.2f,%6.2f> ", v.real(), v.imag());
  };
  for (int i = 0; i < batch_size; i++) {
    for (int j = 0; j < n; j++) {
      data_t result = B[i * stride_b + j];
      data_t residual = result - X[j + i * m];
      passed = passed and (result == result) and
               (std::sqrt(std::abs(std::real(residual * residual))) < bound);
      print(result);
    }
    printf("\n");
  }

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(ptrA_array_dev);
  hipFree(ptrB_array_dev);
  status = hipblasDestroy(h);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("> ERROR: cuBLAS uninitialization failed..\n");
  }

  free(ptrA_array);
  free(ptrB_array);

  if (passed) {
    printf("Calculations successfully finished\n");
  } else {
    printf("ERROR: results mismatch!\n");
    printf("Expected:\n");
    for (int i = 0; i < batch_size; i++) {
      for (int j = 0; j < m; j++) {
        print(X[j + i * m]);
      }
      printf("\n");
    }
    return 1;
  }

  return 0;
}

//
// Description of example setup, APIs used and supported floating point type
// precisions
//
void print_info() {
  std::cout << "" << std::endl;
  std::cout << "########################################################################" << std::endl;
  std::cout << "# Batched strided GELS example:" << std::endl;
  std::cout << "# " << std::endl;
  std::cout << "# Computes least squares of a batch of matrices and right hand sides." << std::endl;
  std::cout << "# Supported floating point type precisions:" << std::endl;
  std::cout << "#   float" << std::endl;
  std::cout << "#   double" << std::endl;
  std::cout << "#   std::complex<float>" << std::endl;
  std::cout << "#   std::complex<double>" << std::endl;
  std::cout << "# " << std::endl;
  std::cout << "########################################################################" << std::endl;
  std::cout << std::endl;
}

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  print_info();

  bool failed = false;

  std::cout << "Running with single precision real data type:" << std::endl;
  failed |= run_gels_batch_example<float>(repeat);

  std::cout << "Running with single precision complex data type:" << std::endl;
  failed |= run_gels_batch_example<std::complex<float>>(repeat);

  std::cout << "Running with double precision real data type:" << std::endl;
  failed |= run_gels_batch_example<double>(repeat);

  std::cout << "Running with double precision complex data type:" << std::endl;
  failed |= run_gels_batch_example<std::complex<double>>(repeat);

  return failed;
}
