#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ 
void rotate (const int n, const float angle, const float3 w, float3 *d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  float s, c;
  sincosf(angle, &s,&c);
  
  const float3 p = d[i];
  const float mc = 1.f - c;

  // Rodrigues' formula:
  float m1 = c+(w.x)*(w.x)*(mc);
  float m2 = (w.z)*s+(w.x)*(w.y)*(mc);
  float m3 =-(w.y)*s+(w.x)*(w.z)*(mc);
  
  float m4 =-(w.z)*s+(w.x)*(w.y)*(mc);
  float m5 = c+(w.y)*(w.y)*(mc);
  float m6 = (w.x)*s+(w.y)*(w.z)*(mc);
  
  float m7 = (w.y)*s+(w.x)*(w.z)*(mc);
  float m8 =-(w.x)*s+(w.y)*(w.z)*(mc);
  float m9 = c+(w.z)*(w.z)*(mc);

  float ox = p.x*m1 + p.y*m2 + p.z*m3;
  float oy = p.x*m4 + p.y*m5 + p.z*m6;
  float oz = p.x*m7 + p.y*m8 + p.z*m9;
  d[i] = {ox, oy, oz};
}

__global__ 
void rotate2 (const int n, const float angle, const float3 w, float4 *d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  float s, c;
  sincosf(angle, &s,&c);
  
  const float4 p = d[i];
  const float mc = 1.f - c;

  // Rodrigues' formula:
  float m1 = c+(w.x)*(w.x)*(mc);
  float m2 = (w.z)*s+(w.x)*(w.y)*(mc);
  float m3 =-(w.y)*s+(w.x)*(w.z)*(mc);
  
  float m4 =-(w.z)*s+(w.x)*(w.y)*(mc);
  float m5 = c+(w.y)*(w.y)*(mc);
  float m6 = (w.x)*s+(w.y)*(w.z)*(mc);
  
  float m7 = (w.y)*s+(w.x)*(w.z)*(mc);
  float m8 =-(w.x)*s+(w.y)*(w.z)*(mc);
  float m9 = c+(w.z)*(w.z)*(mc);

  float ox = p.x*m1 + p.y*m2 + p.z*m3;
  float oy = p.x*m4 + p.y*m5 + p.z*m6;
  float oz = p.x*m7 + p.y*m8 + p.z*m9;
  d[i] = {ox, oy, oz, 0.f};
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <number of points> <repeat>\n", argv[0]);
    return 1;
  }
  const int n = atoi(argv[1]);
  const int repeat = atoi(argv[2]);
    
  // axis of rotation
  const float wx = -0.3, wy = -0.6, wz = 0.15;
  const float norm = 1.f / sqrtf(wx*wx + wy*wy + wz*wz);
  const float3 w = make_float3(wx*norm, wy*norm, wz*norm);

  float angle = 0.5f;

  float3 *h = (float3*) malloc (sizeof(float3) * n);
  float4 *h2 = (float4*) malloc (sizeof(float4) * n);

  srand(123);
  for (int i = 0; i < n; i++) {
    float a = rand();
    float b = rand();
    float c = rand();
    float d = sqrtf(a*a + b*b + c*c);
    h[i] = make_float3(a/d, b/d, c/d);
    h2[i] = make_float4(a/d, b/d, c/d, 0.f);
  }

  dim3 grids ((n + 255) / 256);
  dim3 blocks (256);
 
  float3 *d;
  hipMalloc((void**)&d, sizeof(float3) * n);
  hipMemcpy(d, h, sizeof(float3) * n, hipMemcpyHostToDevice);

  float4 *d2;
  hipMalloc((void**)&d2, sizeof(float4) * n);
  hipMemcpy(d2, h2, sizeof(float4) * n, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    hipLaunchKernelGGL(rotate, grids, blocks, 0, 0, n, angle, w, d);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (float3): %f (us)\n", (time * 1e-3f) / repeat);

  hipDeviceSynchronize();
  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    hipLaunchKernelGGL(rotate2, grids, blocks, 0, 0, n, angle, w, d2);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time (float4): %f (us)\n", (time * 1e-3f) / repeat);

  hipFree(d);
  hipFree(d2);
  free(h);
  free(h2);
  return 0;
}
