#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h> 
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include "utils.h"

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int main(int argc, char *argv[])
{
  int repeat = 1;

  if (argc != 8) {
    printf("Single-precision batched dense matrix - dense matrix multiplication into sparse matrix,\n");
    printf("where the sparse matrix is represented in CSR (Compressed Sparse Row) storage format\n");
    printf("Usage %s <number of batches> <M> <K> <N> <number of non-zero elements> <repeat> <verify>\n", argv[0]);
    printf("SDDM (A, B, C) where (A: M * K, B: K * N, C: M * N)\n");
    return 1;
  }

  int b, m, k, n, nnz, verify;

  b = atoi(argv[1]); // number of batches
  m = atoi(argv[2]);
  k = atoi(argv[3]);
  n = atoi(argv[4]);
  nnz = atoi(argv[5]);
  repeat = atoi(argv[6]);
  verify = atoi(argv[7]);

  const int A_num_rows   = m;
  const int A_num_cols   = k;
  const int B_num_rows   = A_num_cols;
  const int B_num_cols   = n;
  const int C_nnz     = nnz;
  const int lda       = A_num_cols;
  const int ldb       = B_num_cols;
  const int A_size    = lda * A_num_rows;
  const int B_size    = ldb * B_num_rows;
  const int C_size    = A_num_rows * B_num_cols;

  const size_t value_size_bytes  = b * C_nnz * sizeof(float);
  const size_t colidx_size_bytes = b * C_nnz * sizeof(int);
  const size_t rowidx_size_bytes = b * (A_num_rows + 1) * sizeof(size_t);

  float *hA = (float*) malloc (b * A_size * sizeof(float));
  float *hB = (float*) malloc (b * B_size * sizeof(float));

  // contain the result of A * B
  float *hC = (float*) malloc (b * C_size * sizeof(float));

  float *hC_values = (float*) malloc (value_size_bytes);
  int *hC_columns = (int*) malloc (colidx_size_bytes);
  int *hC_offsets = (int*) malloc (rowidx_size_bytes);
  float *hC_result  = (float*) malloc (value_size_bytes);

  for (int i = 0; i < b; i++) {
    init_matrix(hA + i * A_size, A_num_rows, A_num_cols, A_size);
    init_matrix(hB + i * B_size, B_num_rows, B_num_cols, B_size);
    init_matrix(hC + i * C_size, A_num_rows, B_num_cols, C_nnz);

    // initialize the CSR structures for the C matrix
    init_csr(hC_offsets + i * (A_num_rows+1),
             hC_values + i * C_nnz,
             hC_columns + i * C_nnz,
             hC + i * C_size,
             A_num_rows, B_num_cols, C_nnz);
  }

  if (verify) {
    printf("Computing the reference SDDMM results (batch size = %d)..\n", b);
    for (int i = 0; i < b; i++) {
      sddmm (hA + i * A_size,
             hB + i * B_size,
             hC + i * C_size,
             hC_result + i * C_nnz,
             hC_offsets + i * (A_num_rows+1),
             hC_columns + i * C_nnz,
             A_num_cols, A_num_rows, B_num_cols);
    }
    printf("Done\n");
  }

  float alpha        = 1.0f;
  float beta         = 0.0f;
  //--------------------------------------------------------------------------
  // Device memory management
  int *dC_columns;
  size_t *dC_offsets;
  float *dC_values, *dB, *dA;
  CHECK_CUDA( hipMalloc((void**) &dA, b * A_size * sizeof(float)) )
  CHECK_CUDA( hipMalloc((void**) &dB, b * B_size * sizeof(float)) )
  CHECK_CUDA( hipMalloc((void**) &dC_offsets, rowidx_size_bytes) )
  CHECK_CUDA( hipMalloc((void**) &dC_columns, colidx_size_bytes) )
  CHECK_CUDA( hipMalloc((void**) &dC_values,  value_size_bytes) )

  CHECK_CUDA( hipMemcpy(dA, hA, b * A_size * sizeof(float),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dB, hB, b * B_size * sizeof(float),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets, rowidx_size_bytes,
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dC_columns, hC_columns, colidx_size_bytes,
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dC_values, hC_values, value_size_bytes,
                         hipMemcpyHostToDevice) )
  //--------------------------------------------------------------------------
  // CUSPARSE APIs
  hipsparseHandle_t     handle = NULL;
  hipsparseDnMatDescr_t matA, matB;
  hipsparseSpMatDescr_t matC;
  void*                dBuffer    = NULL;
  size_t               bufferSize = 0;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )
  // Create dense matrix A
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matA, b, A_size) )

  // Create dense matrix B
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  CHECK_CUSPARSE( hipsparseDnMatSetStridedBatch(matB, b, B_size) )

  // Create sparse matrix C in CSR format
  CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                    dC_offsets, dC_columns, dC_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
  CHECK_CUSPARSE( hipsparseCsrSetStridedBatch(matC, b, A_num_rows+1, C_nnz) )

  // allocate an external buffer if needed
  CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                               handle,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC, HIP_R_32F,
                               HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
  CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

  // execute preprocess (optional)
  CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
  }
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of SDDMM: %f (us)\n", (time * 1e-3f) / repeat);

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
  CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )

  //--------------------------------------------------------------------------
  if (verify) {
    // device result check
    CHECK_CUDA( hipMemcpy(hC_values, dC_values, value_size_bytes,
                           hipMemcpyDeviceToHost) )
    int correct = 1;

    for (int i = 0; i < b; i++) {
      float *v =  hC_values + i * C_nnz;
      float *r =  hC_result + i * C_nnz;
      for (int j = 0; j < C_nnz; j++) {
        if (fabsf(v[j] - r[j]) > 1e-2f) {
          printf("@batch%d index%d: %f != %f\n", i, j, v[j], r[j]);
          correct = 0;
          break;
        }
      }
      if (!correct) break;
    }
    if (correct)
        printf("sddmm_csr_batched_example test PASSED\n");
    else
        printf("sddmm_csr_batched_example test FAILED: wrong result\n");
  }
  //--------------------------------------------------------------------------
  // device memory deallocation
  CHECK_CUDA( hipFree(dBuffer) )
  CHECK_CUDA( hipFree(dA) )
  CHECK_CUDA( hipFree(dB) )
  CHECK_CUDA( hipFree(dC_offsets) )
  CHECK_CUDA( hipFree(dC_columns) )
  CHECK_CUDA( hipFree(dC_values) )

  free(hA);
  free(hB);
  free(hC);
  free(hC_values);
  free(hC_columns);
  free(hC_offsets);
  free(hC_result);

  return EXIT_SUCCESS;
}
