#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono> // for high_resolution_clock
#include <cstdio>
#include <random>

#include "reference.h"
#include "gpu_solver.h"

void generate_data(int size, int min, int max, float *data) {
  std::mt19937_64 generator{1993764};
  std::uniform_int_distribution<> dist{min, max};
  for (int i = 0; i < size; ++i) {
    data[i] = dist(generator);
  }
}


int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  int repeat = atoi(argv[1]);

  // number of functions
  int N = 1999999;
  printf("N = %d\n", N);

  float *A, *B, *C, *D, *E;
  float *minimum_ref, *minimum;

  checkCuda(hipHostMalloc((void **)&A, N * sizeof(float)));
  checkCuda(hipHostMalloc((void **)&B, N * sizeof(float)));
  checkCuda(hipHostMalloc((void **)&C, N * sizeof(float)));
  checkCuda(hipHostMalloc((void **)&D, N * sizeof(float)));
  checkCuda(hipHostMalloc((void **)&E, N * sizeof(float)));
  checkCuda(hipHostMalloc((void **)&minimum_ref, N * sizeof(float)));
  checkCuda(hipHostMalloc((void **)&minimum, N * sizeof(float)));

  printf("generating data...\n");

  generate_data(N, -100, 100, A);
  generate_data(N, -100, 100, B);
  generate_data(N, -100, 100, C);
  generate_data(N, -100, 100, D);
  generate_data(N, -100, 100, E);

  for (int i = 0; i < N; i++) {
    if (A[i] == 0) {
      A[i] = 1;
    } // avoid undefined behaviour in solver when A=0
  }

  float dur = 0;
  float avg = 0;
  bool ok;

  printf("####################### Reference #############\n");

  for (int k = 0; k < repeat; ++k) {
    auto start = std::chrono::high_resolution_clock::now();

    QuarticMinimumCPU(N, A, B, C, D, E, minimum_ref);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = end - start;
    dur = elapsed.count() * 1000;
    // printf("Time (ms): %f\n", dur);
    avg += dur;
  }

  printf("Execution time (ms): %f\n", avg / repeat);

  avg = 0;

  printf("####################### GPU (no streams) #############\n");

  for (int k = 0; k < repeat; ++k) {

    auto start = std::chrono::high_resolution_clock::now();

    QuarticMinimumGPU(N, A, B, C, D, E, minimum);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = end - start;
    dur = elapsed.count() * 1000;
    // printf("Time (ms): %f\n", dur);
    avg += dur;
  }

  printf("Execution time (ms): %f\n", avg / repeat);

  ok = true;
  for (int i = 0; i < N; i++) {
    if (fabsf(minimum[i] - minimum_ref[i]) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  avg = 0;

  printf("####################### GPU (streams) #############\n");

  for (int k = 0; k < repeat; ++k) {

    auto start = std::chrono::high_resolution_clock::now();

    QuarticMinimumGPUStreams(N, A, B, C, D, E, minimum);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = end - start;
    dur = elapsed.count() * 1000;
    // printf("Time (ms): %f\n", dur);
    avg += dur;
  }

  printf("Execution time (ms): %f\n", avg / repeat);

  ok = true;
  for (int i = 0; i < N; i++) {
    if (fabsf(minimum[i] - minimum_ref[i]) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  checkCuda(hipHostFree(A));
  checkCuda(hipHostFree(B));
  checkCuda(hipHostFree(C));
  checkCuda(hipHostFree(D));
  checkCuda(hipHostFree(E));
  checkCuda(hipHostFree(minimum_ref));
  checkCuda(hipHostFree(minimum));

  return 0;
}
