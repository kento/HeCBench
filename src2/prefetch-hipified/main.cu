#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
 Copyright (c) 2015 - 2021 Advanced Micro Devices, Inc. All rights reserved.
 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:
 The above copyright notice and this permission notice shall be included in
 all copies or substantial portions of the Software.
 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 THE SOFTWARE.
*/

#include <stdio.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define CUDACHECK(error)                                                                       \
{                                                                                              \
    hipError_t localError = error;                                                            \
    if (localError != hipSuccess) {                                                           \
        printf("error: %s at %s:%d\n", hipGetErrorString(localError),  __FILE__, __LINE__);   \
    }                                                                                          \
}

__global__
void add(int n, const float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] += x[i];
}

void prefetch (const int gpuDeviceId, const int numElements, const int repeat)
{
  printf("Concurrent managed access with prefetch\n");

  float *A, *B;

  CUDACHECK(hipMallocManaged(&A, numElements*sizeof(float)));
  CUDACHECK(hipMallocManaged(&B, numElements*sizeof(float)));

  for (int i = 0; i < numElements; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  CUDACHECK(hipDeviceSynchronize());

  float maxError = 0.0f;

  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {

    //CUDACHECK(hipMemAdvise(A, numElements*sizeof(float), hipMemAdviseSetReadMostly, hipCpuDeviceId));
    CUDACHECK(hipMemPrefetchAsync(A, numElements*sizeof(float), gpuDeviceId));
    CUDACHECK(hipMemPrefetchAsync(B, numElements*sizeof(float), gpuDeviceId));

    add <<< dimGrid, dimBlock >>> (numElements, A, B);

    //CUDACHECK(hipMemPrefetchAsync(B, numElements*sizeof(float), hipCpuDeviceId));
    CUDACHECK(hipDeviceSynchronize());
  }

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (ms)\n", time * 1e-6f / repeat);

  for (int i = 0; i < numElements; i++)
    maxError = fmaxf(maxError, fabsf(B[i]-(repeat+2)));

  CUDACHECK(hipFree(A));
  CUDACHECK(hipFree(B));

  bool testResult = (maxError == 0.0f);
  printf("%s\n", testResult ? "PASS" : "FAIL");
}

void naive (const int numElements, const int repeat)
{
  printf("Concurrent managed access without prefetch\n");

  float *A, *B;

  CUDACHECK(hipMallocManaged(&A, numElements*sizeof(float)));
  CUDACHECK(hipMallocManaged(&B, numElements*sizeof(float)));

  for (int i = 0; i < numElements; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  CUDACHECK(hipDeviceSynchronize());

  float maxError = 0.0f;

  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    add <<< dimGrid, dimBlock >>> (numElements, A, B);

    CUDACHECK(hipDeviceSynchronize());
  }

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (ms)\n", time * 1e-6f / repeat);

  for (int i = 0; i < numElements; i++)
    maxError = fmaxf(maxError, fabsf(B[i]-(repeat+2)));

  CUDACHECK(hipFree(A));
  CUDACHECK(hipFree(B));

  bool testResult = (maxError == 0.0f);
  printf("%s\n", testResult ? "PASS" : "FAIL");
}

int main(int argc, char *argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int p_gpuDevice = 0;
  CUDACHECK(hipSetDevice(p_gpuDevice));
  printf("info: set device to %d\n", p_gpuDevice);

  int concurrentManagedAccess = 0;
  CUDACHECK(hipDeviceGetAttribute(&concurrentManagedAccess,
        hipDeviceAttributeConcurrentManagedAccess,
        p_gpuDevice));
  if(!concurrentManagedAccess) {
    printf("info: concurrent managed access not supported on device %d\n Skipped\n", p_gpuDevice);
    return 0;
  }

  const int numElements = 64 * 1024 * 1024;

  for (int i = 0; i < 10; i++) {
    prefetch(p_gpuDevice, numElements, repeat);
  }

  for (int i = 0; i < 10; i++) {
    naive(numElements, repeat);
  }
  return 0;
}
