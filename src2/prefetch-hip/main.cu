#include "hip/hip_runtime.h"
/*
 Copyright (c) 2015 - 2021 Advanced Micro Devices, Inc. All rights reserved.
 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:
 The above copyright notice and this permission notice shall be included in
 all copies or substantial portions of the Software.
 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 THE SOFTWARE.
*/

#include <stdio.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define HIPCHECK(error)                                                                        \
{                                                                                              \
    hipError_t localError = error;                                                             \
    if (localError != hipSuccess) {                                                            \
        printf("error: %s at %s:%d\n", hipGetErrorString(localError),  __FILE__, __LINE__);    \
    }                                                                                          \
}

__global__
void add(int n, const float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] += x[i];
}

void prefetch (const int gpuDeviceId, const int numElements, const int repeat)
{
  printf("Concurrent managed access with prefetch\n");

  float *A, *B;

  HIPCHECK(hipMallocManaged(&A, numElements*sizeof(float)));
  HIPCHECK(hipMallocManaged(&B, numElements*sizeof(float)));

  for (int i = 0; i < numElements; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  HIPCHECK(hipDeviceSynchronize());

  float maxError = 0.0f;

  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {

    //HIPCHECK(hipMemAdvise(A, numElements*sizeof(float), hipMemAdviseSetReadMostly, hipCpuDeviceId));
    HIPCHECK(hipMemPrefetchAsync(A, numElements*sizeof(float), gpuDeviceId));
    HIPCHECK(hipMemPrefetchAsync(B, numElements*sizeof(float), gpuDeviceId));

    hipLaunchKernelGGL(add, dimGrid, dimBlock, 0, 0, numElements, A, B);

    //HIPCHECK(hipMemPrefetchAsync(B, numElements*sizeof(float), hipCpuDeviceId));
    HIPCHECK(hipDeviceSynchronize());
  }

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (ms)\n", time * 1e-6f / repeat);

  for (int i = 0; i < numElements; i++)
    maxError = fmaxf(maxError, fabsf(B[i]-(repeat+2)));

  HIPCHECK(hipFree(A));
  HIPCHECK(hipFree(B));

  bool testResult = (maxError == 0.0f);
  printf("%s\n", testResult ? "PASS" : "FAIL");
}

void naive (const int numElements, const int repeat)
{
  printf("Concurrent managed access without prefetch\n");

  float *A, *B;

  HIPCHECK(hipMallocManaged(&A, numElements*sizeof(float)));
  HIPCHECK(hipMallocManaged(&B, numElements*sizeof(float)));

  for (int i = 0; i < numElements; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  HIPCHECK(hipDeviceSynchronize());

  float maxError = 0.0f;

  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(blockSize, 1, 1);

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    hipLaunchKernelGGL(add, dimGrid, dimBlock, 0, 0, numElements, A, B);

    HIPCHECK(hipDeviceSynchronize());
  }

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time: %f (ms)\n", time * 1e-6f / repeat);

  for (int i = 0; i < numElements; i++)
    maxError = fmaxf(maxError, fabsf(B[i]-(repeat+2)));

  HIPCHECK(hipFree(A));
  HIPCHECK(hipFree(B));

  bool testResult = (maxError == 0.0f);
  printf("%s\n", testResult ? "PASS" : "FAIL");
}

int main(int argc, char *argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int p_gpuDevice = 0;
  HIPCHECK(hipSetDevice(p_gpuDevice));
  printf("info: set device to %d\n", p_gpuDevice);

  int concurrentManagedAccess = 0;
  HIPCHECK(hipDeviceGetAttribute(&concurrentManagedAccess,
        hipDeviceAttributeConcurrentManagedAccess,
        p_gpuDevice));
  if(!concurrentManagedAccess) {
    printf("info: concurrent managed access not supported on device %d\n Skipped\n", p_gpuDevice);
    return 0;
  }

  const int numElements = 64 * 1024 * 1024;

  for (int i = 0; i < 10; i++) {
    prefetch(p_gpuDevice, numElements, repeat);
  }

  for (int i = 0; i < 10; i++) {
    naive(numElements, repeat);
  }
  return 0;
}
