#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <type_traits>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas/hipblas.h>
#include "utils.h"

#define TILE_X 16
#define TILE_Y 16

// M * K, K * N
template <typename T>
__global__ void matrix_mul(T *a, T *b, T *c, int M, int K, int N, T alpha, T beta) {
  int row = blockIdx.y * TILE_Y + threadIdx.y;
  int col = blockIdx.x * TILE_X + threadIdx.x;
  if (row < M && col < N) {
    T s = 0;
    for (int k = 0; k < K; k++)
      s += a[row * K + k] * b[k * N + col];
    c[row * N + col] = alpha * s + beta * c[row * N + col];
  }
}

template <typename T>
void run_simple_gemm(T *a, T *b, T *c, int M, int K, int N, T alpha, T beta) {
  dim3 grids ((N + TILE_X - 1) / TILE_X, (M + TILE_Y - 1) / TILE_Y);
  dim3 blocks (TILE_X, TILE_Y);
  matrix_mul<<<grids, blocks>>>(a, b, c, M, K, N, alpha, beta);
}


//
// Main example for Gemm consisting of
// initialization of A, B and C matrices as well as
// scalars alpha and beta.  Then the product
//
// C = alpha * op(A) * op(B) + beta * C
//
// is performed and finally the results are post processed.
//
template <typename fp>
void run_gemm_example(int m, int k, int n, int repeat) {

  //
  // Initialize data for Gemm
  //
  // C = alpha * op(A) * op(B)  + beta * C
  //

  // set scalar fp values
  const fp alpha = fp(2.0);
  const fp beta  = fp(0.5);

  const size_t A_size = sizeof(fp) * m * k;
  const size_t B_size = sizeof(fp) * k * n;
  const size_t C_size = sizeof(fp) * m * n;

  // prepare matrix data
  fp* a = (fp *) aligned_alloc(64, A_size);
  fp* b = (fp *) aligned_alloc(64, B_size);
  fp* c = (fp *) aligned_alloc(64, C_size);
  fp* r = (fp *) aligned_alloc(64, C_size);

  srand(2);
  rand_matrix(a, m, k);
  rand_matrix(b, k, n);
  rand_matrix(c, m, n);

  fp *da, *db, *dc, *dr;
  hipMalloc((void**)&da, A_size);
  hipMalloc((void**)&db, B_size);
  hipMalloc((void**)&dc, C_size);
  hipMalloc((void**)&dr, C_size);
  hipMemcpy(da, a, A_size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, B_size, hipMemcpyHostToDevice);
  hipMemcpy(dc, c, C_size, hipMemcpyHostToDevice);
  hipMemcpy(dr, c, C_size, hipMemcpyHostToDevice);

  // create execution queue and buffers of matrix data
  hipblasHandle_t h;
  hipblasCreate(&h);

  std::cout << "Checking BLAS GEMM.. ";
  run_simple_gemm(da, db, dr, m, k, n, alpha, beta);

  if constexpr (std::is_same_v<fp, __half>)
    hipblasHgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                &alpha, db, n, da, k, &beta, dc, n);
  else if constexpr (std::is_same_v<fp, float>)
    hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                &alpha, db, n, da, k, &beta, dc, n);
  else if constexpr (std::is_same_v<fp, double>)
    hipblasDgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                &alpha, db, n, da, k, &beta, dc, n);

  hipMemcpy(c, dc, C_size, hipMemcpyDeviceToHost);
  hipMemcpy(r, dr, C_size, hipMemcpyDeviceToHost);
  int error = memcmp(c, r, C_size);
  std::cout << (error ? "FAIL" : "PASS") << std::endl;

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    if constexpr (std::is_same_v<fp, __half>)
      hipblasHgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                  &alpha, db, n, da, k, &beta, dc, n);
    else if constexpr (std::is_same_v<fp, float>)
      hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                  &alpha, db, n, da, k, &beta, dc, n);
    else if constexpr (std::is_same_v<fp, double>)
      hipblasDgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                  &alpha, db, n, da, k, &beta, dc, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  performance(m, n, k, false, time / repeat);

  //
  // Post Processing
  //

#ifdef DEBUG
  std::cout << "\n\t\tOutputting 2x2 block of A,B,C matrices:" << std::endl;

  // output the top 2x2 block of A matrix
  print_2x2_matrix_values(a, k, "A");

  // output the top 2x2 block of B matrix
  print_2x2_matrix_values(b, n, "B");

  // output the top 2x2 block of C matrix
  hipMemcpy(c, dc, C_size, hipMemcpyDeviceToHost);
  print_2x2_matrix_values(c, n, "C");
#endif

  hipblasDestroy(h);

  hipFree(da);
  hipFree(db);
  hipFree(dc);
  hipFree(dr);

  free(a);
  free(b);
  free(c);
  free(r);
}

//
// Main entry point for example.
//
int main (int argc, char ** argv) {
  if (argc != 5) {
    printf("Usage: %s <m> <k> <n> <repeat>\n", argv[0]);
    return 1;
  }
  const int m = atoi(argv[1]);
  const int k = atoi(argv[2]);
  const int n = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  std::cout << "\tRunning with half precision data type:" << std::endl;
  run_gemm_example<__half>(m, k, n, repeat);

  std::cout << "\tRunning with single precision data type:" << std::endl;
  run_gemm_example<float>(m, k, n, repeat);

  std::cout << "\tRunning with double precision data type:" << std::endl;
  run_gemm_example<double>(m, k, n, repeat);

  return 0;
}
