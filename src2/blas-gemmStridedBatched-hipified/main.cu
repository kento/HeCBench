#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include <cmath>
#include <iostream>
#include <type_traits> // is_same
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "reference.h"

using namespace std;

template <typename T>
void gemmBatched(
  int lower,
  int upper,
  int num,
  int reps,
  int verbose) 
{
  if(verbose) cout << "initializing inputs" << endl;
  size_t matrices_size = upper * upper * num * sizeof(T);
  size_t vectors_size = upper * num * sizeof(T);

  T *matrices = (T*)malloc(matrices_size);
  assert(matrices);

  T *vectors = (T*)malloc(vectors_size);
  assert(vectors);

  T *result = (T*)malloc(vectors_size);
  assert(result);

  T *result_ref = (T*)malloc(vectors_size);
  assert(result_ref);

  srand48(48);
  for(int i = 0; i < num * upper * upper; i++)
    matrices[i] = static_cast<T>(drand48());

  for(int i = 0; i < num * upper; i++)
    vectors[i] = static_cast<T>(drand48());

  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  stat = hipblasCreate(&handle);
  if(stat != HIPBLAS_STATUS_SUCCESS){
    cerr << "cublas init failed" << endl;
    exit(1);
  }

  if(verbose) cout << "allocating device variables" << endl;

  // allocate input space on device
  T *devMatrices;
  cudaStat = hipMalloc((void**)&devMatrices, matrices_size);
  assert(!cudaStat);

  T *devVectors;
  cudaStat = hipMalloc((void**)&devVectors, vectors_size);
  assert(!cudaStat);

  // allocate result space on device
  T *devResult;
  cudaStat = hipMalloc((void**)&devResult, vectors_size);

  assert(!cudaStat);

  if(verbose) cout << "copying data to device" << endl;
  // copy data to device
  cudaStat = 
    hipMemcpy(devMatrices, matrices, matrices_size, hipMemcpyHostToDevice);

  assert(!cudaStat);
  
  cudaStat = 
    hipMemcpy(devVectors, vectors, vectors_size, hipMemcpyHostToDevice);

  assert(!cudaStat);

  int lda = upper, // lda >= max(1,m)
      ldb = upper, // ldb >= max(1,k)
      ldc = upper; // ldc >= max(1,m)

  const T alpha = 1.0f, beta = 0.0f;

  /* perform <num> <size x size> x <size x 1> multiplications 
     with distinct matrices
   */
#define GEMM_BATCHED_PARAMETERS handle,              \
                                HIPBLAS_OP_N,         \
                                HIPBLAS_OP_N,         \
                                m, n, k,             \
                                &alpha,              \
                                devMatrices,         \
                                lda,                 \
                                upper * upper,       \
                                devVectors,          \
                                ldb,                 \
                                upper,               \
                                &beta,               \
                                devResult,           \
                                ldc,                 \
                                upper,               \
                                num

  for(int size = lower; size <= upper; size++){
    if(verbose) cout << "running with <size x size> x <size x 1> " << size << endl;
    double sum = 0.0;
    const int m = size, n = 1, k = size;
    for(int rep = 0; rep <= reps; rep++){
      auto start = std::chrono::steady_clock::now();

      if constexpr (std::is_same_v<T, double>)
        stat = hipblasDgemmStridedBatched(GEMM_BATCHED_PARAMETERS);
      else if constexpr (std::is_same_v<T, float>)
        stat = hipblasSgemmStridedBatched(GEMM_BATCHED_PARAMETERS);
      else if constexpr (std::is_same_v<T, __half>)
        stat = hipblasHgemmStridedBatched(GEMM_BATCHED_PARAMETERS);

      hipDeviceSynchronize();
      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      auto elapsed = time * 1e-3;

      if(stat != HIPBLAS_STATUS_SUCCESS){
        cerr << "cublasXgemmStridedBatched failed" << endl;
        break;
      }

      if (rep != 0) sum += elapsed;
      
      if(verbose)
	cout << "size " << size << ": " << elapsed << " us; " 
	     << elapsed / num << " us per operation" << endl;
    }
    cout << "size " << size << " average execution time: " << sum/reps << " us; "
	 << sum / reps / num << " us per operation; "
         << "floating-point operations per second: ";
    performance(m, n, k, 1e3 * (sum / reps / num));

    // verify double precision operations 
    if constexpr (std::is_same_v<T, double>) {
      hipMemcpy(result, devResult, vectors_size, hipMemcpyDeviceToHost);
      gemmBatched_ref (num, upper, upper, 1, m, k, n, alpha, beta,
                       matrices, lda, vectors, ldb, result_ref, ldc);

      for (int i = 0; i < num; i++) {
      for (int j = 0; j < m; j++) {
        if (abs(result[i*upper+j] - result_ref[i*upper+j]) > 1e-6) {
          cout << "Mismatch at batch index " << i << ": " << result[i*upper+j] << "!="
               << result_ref[i*upper+j] << endl;
          break;
        }
      }}
    }
  }

  hipFree(devMatrices);
  hipFree(devVectors);
  hipFree(devResult);
  hipblasDestroy(handle);

  free(matrices);
  free(vectors);
  free(result);
  free(result_ref);
}

int main(int argc, char ** argv){

  int status;
  int lower = 2;    // lower bound to the matrix dimension
  int upper = 100;  // upper bound to the matrix dimension
  int num = 25000;  // batch size
  int reps = 10;
  int verbose = 0;
  
  while((status = getopt(argc, argv, "l:u:n:r:v")) != -1){
    switch(status){
    case 'l':
      lower = strtoul(optarg, 0, 0);
      break;
    case 'u':
      upper = strtoul(optarg, 0, 0);
      break;
    case 'n':
      num = strtoul(optarg, 0, 0);  // batch size
      break;
    case 'r':
      reps = strtoul(optarg, 0, 0);
      break;
    case 'v':
      verbose = 1;
      break;
    default:
      cerr << "invalid argument: " << status << endl;
      exit(1);
    }
  }

  cout << "running with" << " lower: " << lower << " upper: " << upper
       << " num: " << num << " reps: " << reps << endl;

  cout << ">>>>>>>>>>>>>>> Half precision gemmBatched >>>>>>>>>>>>>>> " << endl;
  gemmBatched<__half>(lower, upper, num, reps, verbose);
  cout << ">>>>>>>>>>>>>>> Single precision gemmBatched >>>>>>>>>>>>>>> " << endl;
  gemmBatched<float>(lower, upper, num, reps, verbose);
  cout << ">>>>>>>>>>>>>>> Double precision gemmBatched >>>>>>>>>>>>>>> " << endl;
  gemmBatched<double>(lower, upper, num, reps, verbose);
      
  return 0;
}
