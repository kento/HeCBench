#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This example demonstrates how to use the hipBLAS library API
 * for lower-upper (LU) decomposition of a matrix. LU decomposition
 * factors a matrix as the product of upper triangular matrix and
 * lower trianglular matrix.
 *
 * https://en.wikipedia.org/wiki/LU_decomposition
 *
 * This sample uses 10000 matrices of size NxN and performs
 * LU decomposition of them using batched decomposition API
 * of hipBLAS library. To test the correctness of upper and lower
 * matrices generated, they are multiplied and compared with the
 * original input matrix.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <chrono>

// rocm libraries and helpers
#include <hipblas/hipblas.h>
#include <hip/hip_runtime.h>

#ifndef checkHipErrors
#define checkHipErrors(err)  __checkHipErrors (err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
inline void __checkHipErrors(hipError_t err, const char *file, const int line)
{
  if (hipSuccess != err)
  {
    printf("%s from file %s, line %d\n", hipGetErrorString(err), file, line);
  }
}
#endif

// configurable parameters
// dimension of matrix
#define N 48
#define BATCH_SIZE 10000

/* comment this to use single precision */
//#define DOUBLE_PRECISION 

#ifdef DOUBLE_PRECISION
#define DATA_TYPE double
#define MAX_ERROR 1e-15
#else
#define DATA_TYPE float
#define MAX_ERROR 1e-6
#endif /* DOUBLE_PRCISION */

// wrapper
hipblasStatus_t getrfBatched(hipblasHandle_t handle, int n,
                                   DATA_TYPE* const A[], int lda, int* P,
                                   int* info, int batchSize) {
#ifdef DOUBLE_PRECISION
  return hipblasDgetrfBatched(handle, n, A, lda, P, info, batchSize);
#else
  return hipblasSgetrfBatched(handle, n, A, lda, P, info, batchSize);
#endif
}

// wrapper around malloc
// clears the allocated memory to 0
// terminates the program if malloc fails
void* xmalloc(size_t size) {
  void* ptr = malloc(size);
  if (ptr == NULL) {
    printf("> ERROR: malloc for size %zu failed..\n", size);
    exit(EXIT_FAILURE);
  }
  memset(ptr, 0, size);
  return ptr;
}

// initalize identity matrix
void initIdentityMatrix(DATA_TYPE* mat) {
  // clear the matrix
  memset(mat, 0, N * N * sizeof(DATA_TYPE));

  // set all diagonals to 1
  for (int i = 0; i < N; i++) {
    mat[(i * N) + i] = 1.0;
  }
}

// initialize matrix with all elements as 0
void initZeroMatrix(DATA_TYPE* mat) {
  memset(mat, 0, N * N * sizeof(DATA_TYPE));
}

// fill random value in column-major matrix
void initRandomMatrix(DATA_TYPE* mat) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      mat[(j * N) + i] =
          (DATA_TYPE)1.0 + ((DATA_TYPE)rand() / (DATA_TYPE)RAND_MAX);
    }
  }

  // diagonal dominant matrix to insure it is invertible matrix
  for (int i = 0; i < N; i++) {
    mat[(i * N) + i] += (DATA_TYPE)N;
  }
}

// print column-major matrix
void printMatrix(DATA_TYPE* mat) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      printf("%20.16f ", mat[(j * N) + i]);
    }
    printf("\n");
  }
  printf("\n");
}

// matrix mulitplication
void matrixMultiply(DATA_TYPE* res, DATA_TYPE* mat1, DATA_TYPE* mat2) {
  initZeroMatrix(res);

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      for (int k = 0; k < N; k++) {
        res[(j * N) + i] += mat1[(k * N) + i] * mat2[(j * N) + k];
      }
    }
  }
}

// check matrix equality
bool checkRelativeError(DATA_TYPE* mat1, DATA_TYPE* mat2, DATA_TYPE maxError) {
  DATA_TYPE err = (DATA_TYPE)0.0;
  DATA_TYPE refNorm = (DATA_TYPE)0.0;
  DATA_TYPE relError = (DATA_TYPE)0.0;
  DATA_TYPE relMaxError = (DATA_TYPE)0.0;

  for (int i = 0; i < N * N; i++) {
    refNorm = abs(mat1[i]);
    err = abs(mat1[i] - mat2[i]);

    if (refNorm != 0.0 && err > 0.0) {
      relError = err / refNorm;
      relMaxError = relMaxError > relError ? relMaxError : relError;
    }

    if (relMaxError > maxError) return false;
  }
  return true;
}

// decode lower and upper matrix from single matrix
// returned by getrfBatched()
void getLUdecoded(DATA_TYPE* mat, DATA_TYPE* L, DATA_TYPE* U) {
  // init L as identity matrix
  initIdentityMatrix(L);

  // copy lower triangular values from mat to L (skip diagonal)
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < i; j++) {
      L[(j * N) + i] = mat[(j * N) + i];
    }
  }

  // init U as all zero
  initZeroMatrix(U);

  // copy upper triangular values from mat to U
  for (int i = 0; i < N; i++) {
    for (int j = i; j < N; j++) {
      U[(j * N) + i] = mat[(j * N) + i];
    }
  }
}

// generate permutation matrix from pivot vector
void getPmatFromPivot(DATA_TYPE* Pmat, int* P) {
  int pivot[N];

  // pivot vector in base-1
  // convert it to base-0
  for (int i = 0; i < N; i++) {
    P[i]--;
  }

  // generate permutation vector from pivot
  // initialize pivot with identity sequence
  for (int k = 0; k < N; k++) {
    pivot[k] = k;
  }

  // swap the indices according to pivot vector
  for (int k = 0; k < N; k++) {
    int q = P[k];

    // swap pivot(k) and pivot(q)
    int s = pivot[k];
    int t = pivot[q];
    pivot[k] = t;
    pivot[q] = s;
  }

  // generate permutation matrix from pivot vector
  initZeroMatrix(Pmat);
  for (int i = 0; i < N; i++) {
    int j = pivot[i];
    Pmat[(j * N) + i] = (DATA_TYPE)1.0;
  }
}

int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  // hipBLAS variables
  hipblasStatus_t status;
  hipblasHandle_t handle;

  // host variables
  size_t matSize = N * N * sizeof(DATA_TYPE);

  DATA_TYPE* h_AarrayInput;
  DATA_TYPE* h_AarrayOutput;
  DATA_TYPE* h_ptr_array[BATCH_SIZE];

  int* h_pivotArray;
  int* h_infoArray;

  // device variables
  DATA_TYPE* d_Aarray;
  DATA_TYPE** d_ptr_array;

  int* d_pivotArray;
  int* d_infoArray;

  int err_count = 0;

  // seed the rand() function with time
  srand(12345);

  // initialize hipBLAS
  printf("> initializing..\n");
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("> ERROR: hipBLAS initialization failed..\n");
    return (EXIT_FAILURE);
  }

#ifdef DOUBLE_PRECISION
  printf("> using DOUBLE precision..\n");
#else
  printf("> using SINGLE precision..\n");
#endif

  printf("> pivot ENABLED..\n");

  // allocate memory for host variables
  h_AarrayInput = (DATA_TYPE*)xmalloc(BATCH_SIZE * matSize);
  h_AarrayOutput = (DATA_TYPE*)xmalloc(BATCH_SIZE * matSize);

  h_pivotArray = (int*)xmalloc(N * BATCH_SIZE * sizeof(int));
  h_infoArray = (int*)xmalloc(BATCH_SIZE * sizeof(int));

  // allocate memory for device variables
  checkHipErrors(hipMalloc((void**)&d_Aarray, BATCH_SIZE * matSize));
  checkHipErrors(
      hipMalloc((void**)&d_pivotArray, N * BATCH_SIZE * sizeof(int)));
  checkHipErrors(hipMalloc((void**)&d_infoArray, BATCH_SIZE * sizeof(int)));
  checkHipErrors(
      hipMalloc((void**)&d_ptr_array, BATCH_SIZE * sizeof(DATA_TYPE*)));

  // fill matrix with random data
  printf("> generating random matrices..\n");
  for (int i = 0; i < BATCH_SIZE; i++) {
    initRandomMatrix(h_AarrayInput + (i * N * N));
  }

  // create pointer array for matrices
  for (int i = 0; i < BATCH_SIZE; i++) h_ptr_array[i] = d_Aarray + (i * N * N);

  // copy pointer array to device memory
  checkHipErrors(hipMemcpy(d_ptr_array, h_ptr_array,
                             BATCH_SIZE * sizeof(DATA_TYPE*),
                             hipMemcpyHostToDevice));

  long time = 0;
  // perform LU decomposition
  printf("> performing batched LU decomposition..\n");
  for (int i = 0; i <= repeat; i++) {
    // copy data to device from host
    //printf("> copying data from host memory to GPU memory..\n");
    checkHipErrors(hipMemcpy(d_Aarray, h_AarrayInput, BATCH_SIZE * matSize,
                               hipMemcpyHostToDevice));

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();
    status = getrfBatched(handle, N, d_ptr_array, N, d_pivotArray,
                          d_infoArray, BATCH_SIZE);
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    if (i != 0)
      time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  }

  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("> ERROR: hipblasDgetrfBatched() failed with error %s..\n",
           hipblasStatusToString(status));
  } else {
    printf("Average kernel execution time : %f (us)\n", (time * 1e-3f) / repeat);
  }

  // copy data to host from device
  //printf("> copying data from GPU memory to host memory..\n");
  checkHipErrors(hipMemcpy(h_AarrayOutput, d_Aarray, BATCH_SIZE * matSize,
                           hipMemcpyDeviceToHost));
  checkHipErrors(hipMemcpy(h_infoArray, d_infoArray, BATCH_SIZE * sizeof(int),
                           hipMemcpyDeviceToHost));
  checkHipErrors(hipMemcpy(h_pivotArray, d_pivotArray,
                           N * BATCH_SIZE * sizeof(int),
                           hipMemcpyDeviceToHost));

  // verify the result
  printf("> verifying the result..\n");
  for (int i = 0; i < BATCH_SIZE; i++) {
    if (h_infoArray[i] == 0) {
      DATA_TYPE* A = h_AarrayInput + (i * N * N);
      DATA_TYPE* LU = h_AarrayOutput + (i * N * N);
      DATA_TYPE L[N * N];
      DATA_TYPE U[N * N];
      getLUdecoded(LU, L, U);

      // test P * A = L * U
      int* P = h_pivotArray + (i * N);
      DATA_TYPE Pmat[N * N];
      getPmatFromPivot(Pmat, P);

      // perform matrix multiplication
      DATA_TYPE PxA[N * N];
      DATA_TYPE LxU[N * N];
      matrixMultiply(PxA, Pmat, A);
      matrixMultiply(LxU, L, U);

      // check for equality of matrices
      if (!checkRelativeError(PxA, LxU, (DATA_TYPE)MAX_ERROR)) {
        printf("> ERROR: accuracy check failed for matrix number %05d..\n",
               i + 1);
        err_count++;
      }

    } else if (h_infoArray[i] > 0) {
      printf(
          "> execution for matrix %05d is successful, but U is singular and "
          "U(%d,%d) = 0..\n",
          i + 1, h_infoArray[i] - 1, h_infoArray[i] - 1);
    } else  // (h_infoArray[i] < 0)
    {
      printf("> ERROR: matrix %05d have an illegal value at index %d = %lf..\n",
             i + 1, -h_infoArray[i],
             *(h_AarrayInput + (i * N * N) + (-h_infoArray[i])));
    }
  }

  // free device variables
  checkHipErrors(hipFree(d_ptr_array));
  checkHipErrors(hipFree(d_infoArray));
  checkHipErrors(hipFree(d_pivotArray));
  checkHipErrors(hipFree(d_Aarray));

  // free host variables
  if (h_infoArray) free(h_infoArray);
  if (h_pivotArray) free(h_pivotArray);
  if (h_AarrayOutput) free(h_AarrayOutput);
  if (h_AarrayInput) free(h_AarrayInput);

  // destroy hipBLAS handle
  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("> ERROR: hipBLAS uninitialization failed..\n");
    return (EXIT_FAILURE);
  }

  if (err_count > 0) {
    printf("> TEST FAILED for %d matrices, with precision: %g\n", err_count,
           MAX_ERROR);
    return (EXIT_FAILURE);
  }

  printf("> TEST SUCCESSFUL, with precision: %g\n", MAX_ERROR);
  return (EXIT_SUCCESS);
}
