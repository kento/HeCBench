#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include "utils.h"

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int main(int argc, char *argv[])
{
  int repeat = 1;

  if (argc != 5) {
    printf("The function computes the sum of a sparse vector and a dense vector "); 
    printf("in single-precision floating-point operations\n");
    printf("for i=0 to n-1        \n");
    printf("    Y[i] = beta * Y[i]\n");
    printf("for i=0 to nnz-1      \n");
    printf("    Y[X_indices[i]] += alpha * X_values[i]\n");
    printf("\n");
    printf("Usage %s <M> <N> <nnz> <repeat>\n", argv[0]);
    printf("The size of the vector (n) is M * N\n");
    printf("nnz is the number of non-zero elements\n");
    return 1;
  }

  size_t m, n, nnz;

  m = atol(argv[1]);
  n = atol(argv[2]);
  nnz = atol(argv[3]);
  repeat = atoi(argv[4]);

  // Host problem definition
  const size_t size = m * n;

  const size_t value_size_bytes = nnz * sizeof(float);
  const size_t index_size_bytes = nnz * sizeof(size_t);
  const size_t size_bytes = size * sizeof(size_t);

  float *hA = (float*) malloc (size_bytes);
  float *hB = (float*) malloc (size_bytes); // the initial and final results computed on a host
  float *hY = (float*) malloc (size_bytes); // the results from a device
  float *hA_values = (float*) malloc (value_size_bytes);
  size_t *hA_indices = (size_t*) malloc (index_size_bytes);

  printf("Initializing input matrices..\n");
  init_matrix(hA, m, n, nnz);

  size_t k = 0;
  for (size_t i = 0; i < size; i++) {
    if (hA[i] != 0) {
      hA_indices[k] = i;
      hA_values[k] = hA[i];
      k++;
    }
  }

  init_matrix(hB, m, n, size);

  printf("Done\n");

  const float alpha = 1.0f;
  const float beta  = 1.0f;

  size_t *dX_indices;
  float *dY, *dX_values;
  CHECK_CUDA( hipMalloc((void**) &dX_indices, index_size_bytes)  )
  CHECK_CUDA( hipMalloc((void**) &dX_values,  value_size_bytes)  )
  CHECK_CUDA( hipMalloc((void**) &dY,         size_bytes)  )

  CHECK_CUDA( hipMemcpy(dX_indices, hA_indices, index_size_bytes,
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dX_values, hA_values, value_size_bytes,
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dY, hB, size_bytes,
                         hipMemcpyHostToDevice) )
  //--------------------------------------------------------------------------
  // CUSPARSE APIs
  hipsparseHandle_t     handle = NULL;
  hipsparseSpVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )
  // Create sparse vector X
  CHECK_CUSPARSE( hipsparseCreateSpVec(&vecX, size, nnz, dX_indices, dX_values,
                                      HIPSPARSE_INDEX_64I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
  // Create dense vector y
  CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, size, dY, HIP_R_32F) )

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    // execute Axpby
    CHECK_CUSPARSE( hipsparseAxpby(handle, &alpha, vecX, &beta, vecY) )
  }
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of SPAXPBY : %f (us)\n", (time * 1e-3f) / repeat);

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE( hipsparseDestroySpVec(vecX) )
  CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )
  //--------------------------------------------------------------------------
  // device result check
  CHECK_CUDA( hipMemcpy(hY, dY, size_bytes,
                         hipMemcpyDeviceToHost) )

  printf("Computing the reference results..\n");
  for (int n = 0; n < repeat; n++) {
    #pragma omp parallel for
    for (size_t i = 0; i < size; i++) {
      hB[i] = alpha * hA[i] + beta * hB[i];
    }
  }
  printf("Done\n");
  
  int correct = 1;
  for (size_t i = 0; i < size; i++) {
    if (fabsf(hY[i] - hB[i]) > 1e-2f) {
      correct = 0;
      break;
    }
  }
  if (correct)
      printf("axpby_example test PASSED\n");
  else
      printf("axpby_example test FAILED: wrong result\n");
  //--------------------------------------------------------------------------
  // device memory deallocation
  CHECK_CUDA( hipFree(dX_indices) )
  CHECK_CUDA( hipFree(dX_values)  )
  CHECK_CUDA( hipFree(dY) )
  free(hA);
  free(hB);
  free(hY);
  free(hA_values);
  free(hA_indices);
  return EXIT_SUCCESS;
}
