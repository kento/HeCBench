#include <chrono>
#include <stdio.h>
#include <hip/hip_runtime.h>

// reference
// https://stackoverflow.com/questions/59879285/whats-the-alternative-for-match-any-sync-on-compute-capability-6

// increment the value at ptr by 1 and return the old value
__device__ int atomicAggInc(int* ptr) {
  int mask;
  unsigned tmask = 0xFFFFFFFF;
#if __CUDA_ARCH__ >= 700
    // return mask of threads that have same value in tmask
    mask = __match_any_sync(tmask, (unsigned long long)ptr);
#else
  for (int i = 0; i < warpSize; i++){
    unsigned long long tptr = __shfl_sync(tmask, (unsigned long long)ptr, i);
    unsigned my_mask = __ballot_sync(tmask, (tptr == (unsigned long long)ptr));
    if (i == (threadIdx.x & (warpSize-1))) mask = my_mask;
  }
#endif
  int leader = __ffs(mask) - 1;  // select a leader
  int res = 0;
  unsigned lane_id = threadIdx.x % warpSize;
  if (lane_id == leader) {                 // leader does the update
    res = atomicAdd(ptr, __popc(mask));
  }
  res = __shfl_sync(mask, res, leader);    // get leader’s old value
  return res + __popc(mask & ((1 << lane_id) - 1)); //compute old value
}

__global__ void k(int *d, int s) {
  int *ptr = d + threadIdx.x % s;
  atomicAggInc(ptr);
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  const int nBlocks = 65536;
  const int blockSize = 256;

  for (int ds = 32; ds >= 1; ds = ds / 2) {
    int *d_d, *h_d;
    h_d = new int[ds];
    hipMalloc((void**)&d_d, ds*sizeof(int));
    hipMemset(d_d, 0, ds*sizeof(int));

    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++)
      k<<<nBlocks, blockSize>>>(d_d, ds);
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<float> time = end - start;
    printf("Total kernel time (%d locations): %f (s)\n", ds, time.count());

    hipMemcpy(h_d, d_d, ds*sizeof(int), hipMemcpyDeviceToHost);

    bool ok = true;
    for (int i = 0; i < ds; i++) {
      if (h_d[i] != blockSize / ds * nBlocks * repeat) {
        ok = false;
        break;
      }
    }
    printf("%s\n", ok ? "PASS" : "FAIL");
    hipFree(d_d);
    delete [] h_d;
  }
  return 0;
}
