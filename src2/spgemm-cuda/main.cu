#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include "utils.h"

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int COO(int m, int k, int n, int a_nnz, int repeat, int verify) {
  // Host problem definition
  const int A_num_rows = m;
  const int A_num_cols = k;
  const int A_nnz      = a_nnz;
  const int B_num_rows = A_num_cols;
  const int B_num_cols = n;
  const int lda        = A_num_cols;
  const int ldb        = B_num_cols;
  const int ldc        = B_num_cols;
  const int A_size     = lda * A_num_rows;
  const int B_size     = ldb * B_num_rows;
  const int C_size     = ldc * A_num_rows;

  float *hA = (float*) malloc (A_size * sizeof(float));
  float *hB = (float*) malloc (B_size * sizeof(float));

  const size_t A_value_size_bytes  = A_nnz * sizeof(float);
  const size_t A_colidx_size_bytes = A_nnz * sizeof(int);
  const size_t A_rowidx_size_bytes = A_nnz * sizeof(int);

  float *hA_values = (float*) malloc (A_value_size_bytes);
  int *hA_columns = (int*) malloc (A_colidx_size_bytes);
  int *hA_rows = (int*) malloc (A_rowidx_size_bytes);

  init_matrix(hA, A_num_rows, A_num_cols, A_nnz);
  init_coo(hA_rows, hA_values, hA_columns, hA,
           A_num_rows, A_num_cols, A_nnz);

  init_matrix(hB, B_num_rows, B_num_cols, B_size);

  float               alpha       = 1.0f;
  float               beta        = 0.0f;
  hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  //--------------------------------------------------------------------------
  // Device memory management: Allocate and copy A, B
  int   *dA_rows, *dA_columns;
  float *dA_values, *dB, *dC;
  // allocate A
  CHECK_CUDA( hipMalloc((void**) &dA_rows, A_nnz * sizeof(int)) )
  CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
  CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float)) )
  // allocate B
  CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(float)) )
  // allocate C
  CHECK_CUDA( hipMalloc((void**) &dC, C_size * sizeof(float)) )

  // copy A
  CHECK_CUDA( hipMemcpy(dA_rows, hA_rows, A_nnz * sizeof(int),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                         A_nnz * sizeof(float), hipMemcpyHostToDevice) )
  // copy B
  CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float), hipMemcpyHostToDevice) )

  //--------------------------------------------------------------------------
  // CUSPARSE APIs
  hipsparseHandle_t     handle = NULL;
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  void*  dBuffer    = NULL;
  size_t bufferSize = 0;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )
  // Create sparse matrix A in COO format
  CHECK_CUSPARSE( hipsparseCreateCoo(&matA, A_num_rows, A_num_cols, A_nnz,
                                    dA_rows, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, B_num_rows, B_num_cols, ldb,
                                      dB, HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc,
                                      dC, HIP_R_32F, HIPSPARSE_ORDER_ROW) )

  // allocate an external buffer if needed
  CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                               handle,
                               opA,
                               opB,
                               &alpha, matA, matB, &beta, matC, HIP_R_32F,
                               HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
  CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {

    // compute the product of A * B
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 opA,
                                 opB,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

  }
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of SPGEMM (COO) compute: %f (us)\n", (time * 1e-3f) / repeat);

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )

  //--------------------------------------------------------------------------
  // device result check
  
  if (verify) {
    printf("Computing the reference SPGEMM results..\n");
    float *hC = (float*) malloc (C_size * sizeof(float));
    gemm (hA, hB, hC, A_num_cols, A_num_rows, B_num_cols);

    float *hC_tmp = (float*) malloc (C_size * sizeof(float));
    CHECK_CUDA( hipMemcpy(hC_tmp, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) )
    int correct = 1;
    for (int i = 0; i < C_size; i++) {
      if (fabsf(hC_tmp[i] - hC[i]) > 1e-2f) {
        printf("@%d %f != %f\n", i, hC_tmp[i], hC[i]);
        correct = 0;                       
        break;
      }
    }

    free(hC_tmp);
    free(hC);

    if (correct)
      printf("spgemm_example test PASSED\n");
    else
      printf("spgemm_example test FAILED: wrong result\n");
  }

  //--------------------------------------------------------------------------
  // device memory deallocation
  CHECK_CUDA( hipFree(dBuffer) )
  CHECK_CUDA( hipFree(dA_rows) )
  CHECK_CUDA( hipFree(dA_columns) )
  CHECK_CUDA( hipFree(dA_values) )
  CHECK_CUDA( hipFree(dB) )
  CHECK_CUDA( hipFree(dC) )
  free(hA);
  free(hB);
  free(hA_values);
  free(hA_columns);
  free(hA_rows);
  return EXIT_SUCCESS;
}

int CSR(int m, int k, int n, int a_nnz, int repeat, int verify) {
  // Host problem definition
  const int A_num_rows = m;
  const int A_num_cols = k;
  const int A_nnz      = a_nnz;
  const int B_num_rows = A_num_cols;
  const int B_num_cols = n;
  const int lda        = A_num_cols;
  const int ldb        = B_num_cols;
  const int ldc        = B_num_cols;
  const int A_size     = lda * A_num_rows;
  const int B_size     = ldb * B_num_rows;
  const int C_size     = ldc * A_num_rows;

  float *hA = (float*) malloc (A_size * sizeof(float));
  float *hB = (float*) malloc (B_size * sizeof(float));

  const size_t A_value_size_bytes  = A_nnz * sizeof(float);
  const size_t A_colidx_size_bytes = A_nnz * sizeof(int);
  const size_t A_rowidx_size_bytes = (A_num_rows + 1) * sizeof(int);

  float *hA_values = (float*) malloc (A_value_size_bytes);
  int *hA_columns = (int*) malloc (A_colidx_size_bytes);
  int *hA_offsets = (int*) malloc (A_rowidx_size_bytes);

  init_matrix(hA, A_num_rows, A_num_cols, A_nnz);
  init_csr(hA_offsets, hA_values, hA_columns, hA,
           A_num_rows, A_num_cols, A_nnz);

  init_matrix(hB, B_num_rows, B_num_cols, B_size);

  float               alpha       = 1.0f;
  float               beta        = 0.0f;
  hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  //--------------------------------------------------------------------------
  // Device memory management: Allocate and copy A, B
  int   *dA_offsets, *dA_columns;
  float *dA_values, *dB, *dC;
  // allocate A
  CHECK_CUDA( hipMalloc((void**) &dA_offsets,
                         (A_num_rows + 1) * sizeof(int)) )
  CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
  CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float)) )
  // allocate B
  CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(float)) )
  // allocate C
  CHECK_CUDA( hipMalloc((void**) &dC, C_size * sizeof(float)) )

  // copy A
  CHECK_CUDA( hipMemcpy(dA_offsets, hA_offsets,
                         (A_num_rows + 1) * sizeof(int),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                         A_nnz * sizeof(float), hipMemcpyHostToDevice) )
  // copy B
  CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float), hipMemcpyHostToDevice) )

  //--------------------------------------------------------------------------
  // CUSPARSE APIs
  hipsparseHandle_t     handle = NULL;
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  void*  dBuffer    = NULL;
  size_t bufferSize = 0;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )
  // Create sparse matrix A in CSR format
  CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                    dA_offsets, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, B_num_rows, B_num_cols, ldb,
                                      dB, HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc,
                                      dC, HIP_R_32F, HIPSPARSE_ORDER_ROW) )

  // allocate an external buffer if needed
  CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                               handle,
                               opA,
                               opB,
                               &alpha, matA, matB, &beta, matC, HIP_R_32F,
                               HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
  CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {

    // compute the product of A * B
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 opA,
                                 opB,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

  }
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of SPGEMM (CSR) compute: %f (us)\n", (time * 1e-3f) / repeat);

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )

  //--------------------------------------------------------------------------
  // device result check
  
  if (verify) {
    printf("Computing the reference SPGEMM results..\n");
    float *hC = (float*) malloc (C_size * sizeof(float));
    gemm (hA, hB, hC, A_num_cols, A_num_rows, B_num_cols);

    float *hC_tmp = (float*) malloc (C_size * sizeof(float));
    CHECK_CUDA( hipMemcpy(hC_tmp, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) )
    int correct = 1;
    for (int i = 0; i < C_size; i++) {
      if (fabsf(hC_tmp[i] - hC[i]) > 1e-2f) {
        printf("@%d %f != %f\n", i, hC_tmp[i], hC[i]);
        correct = 0;                       
        break;
      }
    }

    free(hC_tmp);
    free(hC);

    if (correct)
      printf("spgemm_example test PASSED\n");
    else
      printf("spgemm_example test FAILED: wrong result\n");
  }

  //--------------------------------------------------------------------------
  // device memory deallocation
  CHECK_CUDA( hipFree(dBuffer) )
  CHECK_CUDA( hipFree(dA_offsets) )
  CHECK_CUDA( hipFree(dA_columns) )
  CHECK_CUDA( hipFree(dA_values) )
  CHECK_CUDA( hipFree(dB) )
  CHECK_CUDA( hipFree(dC) )
  free(hA);
  free(hB);
  free(hA_values);
  free(hA_columns);
  free(hA_offsets);
  return EXIT_SUCCESS;
}

int main(int argc, char *argv[])
{
  int repeat = 1;

  if (argc != 7) {
    printf("Single-precision sparse matrix-dense matrix multiplication into dense matrix,\n");
    printf("where the sparse matrix is represented in COO and CSR storage format\n");
    printf("Usage %s <M> <K> <N> <A_nnz> <repeat> <verify>\n", argv[0]);
    printf("SPMM (A, B, C) where (A: M * K, B: K * N, C: M * N)\n");
    return 1;
  }

  int m, k, n, a_nnz, verify;

  m = atoi(argv[1]);
  k = atoi(argv[2]);
  n = atoi(argv[3]);
  a_nnz = atoi(argv[4]);
  repeat = atoi(argv[5]);
  verify = atoi(argv[6]);

  COO(m, k, n, a_nnz, repeat, verify);
  CSR(m, k, n, a_nnz, repeat, verify);

  return EXIT_SUCCESS;
}
