#include "hip/hip_runtime.h"
// *********************************************************************
// A simple demo application that implements a
// vector dot product computation between two arrays.
//
// Runs computations with on the GPU device and then checks results
// *********************************************************************

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hipblas.h>

template <typename T>
void dot (const size_t iNumElements, const int iNumIterations)
{
  const size_t src_size = iNumElements;
  const size_t src_size_bytes = src_size * sizeof(T);

  // Allocate and initialize host arrays
  T* srcA = (T*) malloc (src_size_bytes);
  T* srcB = (T*) malloc (src_size_bytes);
  T  dst;

  size_t i;
  double sum = 0.0;
  for (i = 0; i < iNumElements ; ++i)
  {
    srcA[i] = (T)(sqrt(65504.0 / iNumElements));
    srcB[i] = (T)(sqrt(65504.0 / iNumElements));
    sum += (float)srcA[i] * (float)srcB[i];
  }

  T *d_srcA;
  T *d_srcB;
  T *d_dst;

  hipMalloc((void**)&d_srcA, src_size_bytes);
  hipMemcpy(d_srcA, srcA, src_size_bytes, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_srcB, src_size_bytes);
  hipMemcpy(d_srcB, srcB, src_size_bytes, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dst, sizeof(T));


  hipblasHandle_t h;
  hipblasCreate(&h);
  hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);

  hipDataType xType, yType, rType, eType;
  if constexpr (std::is_same<T, double>::value) {
    xType = yType = rType = eType = HIP_R_64F;
  } else if constexpr (std::is_same<T, float>::value) {
    xType = yType = rType = eType = HIP_R_32F;
  } else if constexpr (std::is_same<T, __half>::value) {
    xType = yType = rType = HIP_R_16F;
    eType = HIP_R_32F;
  } else if constexpr (std::is_same<T, hip_bfloat16>::value) {
    xType = yType = rType = HIP_R_16BF;
    eType = HIP_R_32F;
  }

  auto start = std::chrono::steady_clock::now();

  for (i = 0; i < (size_t)iNumIterations; i++) {
    hipblasDotEx_v2(h, iNumElements, d_srcA, xType, 1, d_srcB,
                yType, 1, d_dst, rType, eType);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average hipblasDotEx_v2 execution time %f (ms)\n", (time * 1e-6f) / iNumIterations);

  hipMemcpy(&dst, d_dst, sizeof(T), hipMemcpyDeviceToHost);
  printf("Host: %lf  Device: %lf\n", sum, double(dst));
  printf("%s\n\n", (fabs(double(dst) - sum) < 1e-1) ? "PASS" : "FAIL");

  hipFree(d_dst);
  hipFree(d_srcA);
  hipFree(d_srcB);
  hipblasDestroy(h);

  free(srcA);
  free(srcB);
}

int main(int argc, char **argv)
{
  if (argc != 3) {
    printf("Usage: %s <number of elements> <repeat>\n", argv[0]);
    return 1;
  }
  const size_t iNumElements = atol(argv[1]);
  const int iNumIterations = atoi(argv[2]);

  printf("\nFP64 Dot\n");
  dot<double>(iNumElements, iNumIterations);
  printf("\nFP32 Dot\n");
  dot<float>(iNumElements, iNumIterations);
  printf("\nFP16 Dot\n");
  dot<__half>(iNumElements, iNumIterations);
  printf("\nBF16 Dot\n");
  dot<hip_bfloat16>(iNumElements, iNumIterations);

  return EXIT_SUCCESS;
}
