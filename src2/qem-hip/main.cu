#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono> // for high_resolution_clock
#include <cstdio>
#include <random>

#include "reference.h"
#include "gpu_solver.h"

void generate_data(int size, int min, int max, float *data) {
  std::mt19937_64 generator{1993764};
  std::uniform_int_distribution<> dist{min, max};
  for (int i = 0; i < size; ++i) {
    data[i] = dist(generator);
  }
}


int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  int repeat = atoi(argv[1]);

  // number of functions
  int N = 1999999;
  printf("N = %d\n", N);

  float *A, *B, *C, *D, *E;
  float *minimum_ref, *minimum;

  checkHip(hipHostMalloc((void **)&A, N * sizeof(float)));
  checkHip(hipHostMalloc((void **)&B, N * sizeof(float)));
  checkHip(hipHostMalloc((void **)&C, N * sizeof(float)));
  checkHip(hipHostMalloc((void **)&D, N * sizeof(float)));
  checkHip(hipHostMalloc((void **)&E, N * sizeof(float)));
  checkHip(hipHostMalloc((void **)&minimum_ref, N * sizeof(float)));
  checkHip(hipHostMalloc((void **)&minimum, N * sizeof(float)));

  printf("generating data...\n");

  generate_data(N, -100, 100, A);
  generate_data(N, -100, 100, B);
  generate_data(N, -100, 100, C);
  generate_data(N, -100, 100, D);
  generate_data(N, -100, 100, E);

  for (int i = 0; i < N; i++) {
    if (A[i] == 0) {
      A[i] = 1;
    } // avoid undefined behaviour in solver when A=0
  }

  float dur = 0;
  float avg = 0;
  bool ok;

  printf("####################### Reference #############\n");

  for (int k = 0; k < repeat; ++k) {
    auto start = std::chrono::high_resolution_clock::now();

    QuarticMinimumCPU(N, A, B, C, D, E, minimum_ref);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = end - start;
    dur = elapsed.count() * 1000;
    // printf("Time (ms): %f\n", dur);
    avg += dur;
  }

  printf("Execution time (ms): %f\n", avg / repeat);

  avg = 0;

  printf("####################### GPU (no streams) #############\n");

  for (int k = 0; k < repeat; ++k) {

    auto start = std::chrono::high_resolution_clock::now();

    QuarticMinimumGPU(N, A, B, C, D, E, minimum);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = end - start;
    dur = elapsed.count() * 1000;
    // printf("Time (ms): %f\n", dur);
    avg += dur;
  }

  printf("Execution time (ms): %f\n", avg / repeat);

  ok = true;
  for (int i = 0; i < N; i++) {
    if (fabsf(minimum[i] - minimum_ref[i]) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  avg = 0;

  printf("####################### GPU (streams) #############\n");

  for (int k = 0; k < repeat; ++k) {

    auto start = std::chrono::high_resolution_clock::now();

    QuarticMinimumGPUStreams(N, A, B, C, D, E, minimum);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = end - start;
    dur = elapsed.count() * 1000;
    // printf("Time (ms): %f\n", dur);
    avg += dur;
  }

  printf("Execution time (ms): %f\n", avg / repeat);
  ok = true;
  for (int i = 0; i < N; i++) {
    if (fabsf(minimum[i] - minimum_ref[i]) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  checkHip(hipHostFree(A));
  checkHip(hipHostFree(B));
  checkHip(hipHostFree(C));
  checkHip(hipHostFree(D));
  checkHip(hipHostFree(E));
  checkHip(hipHostFree(minimum));
  checkHip(hipHostFree(minimum_ref));

  return 0;
}
