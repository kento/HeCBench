#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "grid.h"
#include "constants.h"

struct grid_t init_grid(llint nx, llint ny, llint nz, llint tsx, llint tsy)
{
    struct grid_t grid;
    grid.nx = nx; grid.ny = ny; grid.nz = nz;
    grid.dx = 20;  grid.dy = 20;  grid.dz = 20;
    grid.lx = 4; grid.ly = 4; grid.lz = 4;
    grid.ntaperx = 3; grid.ntapery = 3; grid.ntaperz = 3;

    // extending x-dim with extra two radius just so that boundary checking can be eliminated from the implementation
    grid.ldimx = nx + 4 * grid.lx;
    grid.ldimy = ny + 2 * grid.ly;
    // Padding the Z dimension to align on 128B, for grid size of 1000, this is effectively 1024
    grid.ldimz = ((nz + 2 * grid.lz + 31) / 32) * 32;

    printf("ldimx: %lld, ldimy: %lld, ldimz: %lld\n", grid.ldimx, grid.ldimy, grid.ldimz);

    const float lambdamax = vmax/_fmax;
    grid.ndampx = grid.ntaperx * lambdamax / grid.dx;
    grid.ndampy = grid.ntapery * lambdamax / grid.dy;
    grid.ndampz = grid.ntaperz * lambdamax / grid.dz;

    grid.x1 = 0;
    grid.x2 = grid.ndampx;
    grid.x3 = grid.ndampx;
    grid.x4 = grid.nx-grid.ndampx;
    grid.x5 = grid.nx-grid.ndampx;
    grid.x6 = grid.nx;

    grid.y1 = 0;
    grid.y2 = grid.ndampy;
    grid.y3 = grid.ndampy;
    grid.y4 = grid.ny-grid.ndampy;
    grid.y5 = grid.ny-grid.ndampy;
    grid.y6 = grid.ny;

    grid.z1 = 0;
    grid.z2 = grid.ndampz;
    grid.z3 = grid.ndampz;
    grid.z4 = grid.nz-grid.ndampz;
    grid.z5 = grid.nz-grid.ndampz;
    grid.z6 = grid.nz;

    grid.tsx = tsx;
    grid.tsy = tsy;
    grid.ntx = nx/tsx;
    grid.nty = ny/tsy;

    printf("ndamp = %lld %lld %lld\n", grid.ndampx, grid.ndampy, grid.ndampz);
    return grid;
}

// Lead padding needed to align element (0,0,ndampz) on 128B cache line
size_t getLeadpad (struct grid_t grid)
{
  int align32 = (grid.lz + grid.ndampz) & 31;
  return (align32 ? 32 - align32 : 0);
}

// Useful size of the grid, in bytes
size_t gridSize (struct grid_t grid)
{
  size_t size = grid.ldimx * grid.ldimy * grid.ldimz * sizeof (float);
  return size;
}

// Device grid, with lead padding
float * allocateDeviceGrid (struct grid_t grid)
{
  int leadpad = getLeadpad(grid);
  size_t size = gridSize(grid) + leadpad * sizeof (float);
  float *ptr;
  if (hipMalloc ((void **)&ptr, size) != hipSuccess) {
      fprintf(stderr, "hipMalloc failed!");
      return NULL;
  }
  return (ptr + leadpad);
}

void freeDeviceGrid (float *ptr, struct grid_t grid)
{
  int leadpad = getLeadpad(grid);
  hipFree (ptr - leadpad);
}

// Host grid, with lead padding
float * allocateHostGrid (struct grid_t grid)
{
  int leadpad = getLeadpad(grid);
  size_t size = gridSize(grid) + leadpad * sizeof (float);
  float *ptr;
  if (hipHostMalloc ((void **)&ptr, size) != hipSuccess) {
      fprintf(stderr, "hipHostMalloc failed!");
      return NULL;
  }
  return (ptr + leadpad);
}

void freeHostGrid (float *ptr, struct grid_t grid)
{
  int leadpad = getLeadpad(grid);
  hipHostFree (ptr - leadpad);
}
